#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/sequence.h>
#include <thrust/count.h>

#define PI 3.14159f
#define MAXP 8000
#define MAXN 96
#define MAXG 1280000

#define THREADS 256

struct pair {
	int key;
	int value;
};

struct  grid {
    float oX, oY, oZ;
    float size;
    int nX, nY, nZ;
};

struct  simulation {
    float minX, maxX;
    float minY, maxY;
    float minZ, maxZ;
    float dt;
    int tsn;
    int ssi;
    int nsi;
};

struct load {
    float minX, maxX;
    float minY, maxY;
    float minZ, maxZ;
    float gx;
    float gy;
    float gz;
    float w;
};

struct fix {
    float minX, maxX;
    float minY, maxY;
    float minZ, maxZ;
    float velX, velY, velZ;
};

struct outlet {
    float oX, oY, oZ;
    float nX, nY, nZ;
};

struct inlet {
    int Material;
    float Mass, Smooth;
    float oX, oY, oZ;
    float uX, uY, uZ;
    float vX, vY, vZ;
    float nX, nY, nZ;
    int nu, nv;
    float Velocity;
    float Density, Energy;
    float Distance;
};


struct model {
    int pn;
    int* Material;
    float* Mass;
    float* Smooth;
    float* PosX;
    float* PosY;
    float* PosZ;
    float* VelX;
    float* VelY;
    float* VelZ;
    float* Density;
    float* Energy;
    float* Pressure;
    float* Sound;
    float* VelDotX;
    float* VelDotY;
    float* VelDotZ;
    float* DensityDot;
    float* EnergyDot;
    float* PosX0;
    float* PosY0;
    float* PosZ0;
    float* VelX0;
    float* VelY0;
    float* VelZ0;
    float* Density0;
    float* Energy0;
    int* List;
    int* Hash;
    int* Index;
    int* SetStart;
    int* SetStop;
    int* IntDummy;
    float* FloatDummy;
};


// Host Variables

int *hMaterial;
float *hPosX;
float *hPosY;
float *hVelX;
float *hVelY;
float *hDensity;
float *hEnergy;
float *hPressure;
float *hVelDotX;
float *hVelDotY;
float *hDensityDot;
float *hEnergyDot;
int *hList;
int *hHash;
int *hIndex;
int *hSetStart;
int *hSetStop;

int hPN;
float hSmooth, hMass, hSound;
int hMatType[10];
float hMatProp[10][10];
struct simulation hRun;
struct grid hGrid;
struct load hLoad[10];
struct fix hFix[10];
struct outlet hOut[10];
struct inlet hIn[10];

float *hPosX0;
float *hPosY0;
float *hVelX0;
float *hVelY0;
float *hDensity0;
float *hEnergy0;

// Device Variables
__device__ __constant__ int dMatType[10];
__device__ __constant__ float dMatProp[10][10];
__device__ __constant__ struct simulation dRun;
__device__ struct grid dGrid;
__device__ __constant__ struct load dLoad[10];
__device__ __constant__ struct fix dFix[10];
__device__ __constant__ struct outlet dOut[10];
__device__ struct inlet dIn[10];


__host__ __device__ float kernelWendland(float r, float h) {

    float q, alpha, w;
    /**
     * \brief Wendland kernel
     *
     * \date Feb 8, 2011
     * \author Luca Massidda
     */

    q = r / h;

    // for 3D
    alpha = 15.0f / (16.0f * PI * h * h * h);

    // for 2D
    //alpha = 7.0f / (4.0f * PI * h * h);

    w = 0.0f;
    if (q < 2) {
        w = powf((1.0f - 0.5f*q),4);
        w *= 1.0f + 2.0f*q;
        w *= alpha;
    }

    return w;
}


__host__ __device__ float kernelDerivWendland(float r, float h) {

    float q, alpha, dwdr;
    /**
     * \brief Wendland kernel derivative
     *
     * \date Feb 8, 2011
     * \author Luca Massidda
     */

    q = r / h;

    // for 3D
    alpha = 15.0f / (16.0f * PI * h * h * h);

    // for 2D
    //alpha = 7.0f / (4.0f * PI * h * h);

    dwdr = 0.0f;
    if (q < 2) {
        dwdr = 5.0f / 8.0f * q * powf((q - 2.0f), 3) ;
        dwdr *= alpha / h;
    }

    return dwdr;
}



float pressureGasHost(int mat ,float rho, float u) {
    /**
     * \brief Ideal gas Equation Of State
     *
     * p = (k -1) rho u
     * c = (k(k -1) u)^0.5
     *
     * k = dMatProp[mat][1]
     * pshift = dMatProp[mat][2]
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float p;
//    float c;

    p = (hMatProp[mat][1] - 1.0) * rho * u;
    p += hMatProp[mat][2];

//    c = sqrtf(hMatProp[mat][1] * (hMatProp[mat][1] - 1.0) * u);

    return p;
}



float pressurePolyHost(int mat , float rho, float u) {
    /**
     * \brief Mie-Gruneisen polynomial Equation Of State
     *
     * p = a1 mu + a2 mu^2 + a3 mu^3 + (b0 + b1 mu) rho0 u  in compression
     * p = t1 mu + t2 mu^2 + b0 rho0 u                      in tension
     *
     * rho0 = dMatProp[mat][0];
     * a1 = dMatProp[mat][1];
     * a2 = dMatProp[mat][2];
     * a3 = dMatProp[mat][3];
     * b0 = dMatProp[mat][4];
     * b1 = dMatProp[mat][5];
     * t1 = dMatProp[mat][6];
     * t2 = dMatProp[mat][7];
     * pmin = dMatProp[mat][8];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float mu;
    float p;
//    float c;

    mu = (rho - hMatProp[mat][0]) / hMatProp[mat][0];

    if (mu < 0)
        p = (hMatProp[mat][6] * mu + hMatProp[mat][7] * mu*mu)
            + (hMatProp[mat][4] * hMatProp[mat][0] * u);
    else
        p = (hMatProp[mat][1] * mu + hMatProp[mat][2] * mu*mu
             + hMatProp[mat][3] * mu*mu*mu)
            + ((hMatProp[mat][4] + hMatProp[mat][5] * mu)
               * hMatProp[mat][0] * u);

    if (p < hMatProp[mat][8]) p = hMatProp[mat][8];

//    c = sqrtf(hMatProp[mat][1] / rho);

    return p;
}

float pressureShockHost(int mat, float rho, float u) {
    /**
     * \brief Mie-Gruneisen Shock Hugoniot Equation Of State
     *
     * mu = rho / rho0 -1
     * g = g * rho0 / rho
     * ph = (rho0 c0^2 mu (1 + mu)) / (1 - (s0 - 1) * mu)^2
     * uh = 1/2 ph/rho0 * (mu / (1 + mu))
     * p = ph + g * rho * (u - uh)
     *
     * rho0 = dMatProp[mat][0];
     * c0 = dMatProp[mat][1];
     * g0 = dMatProp[mat][2];
     * s0 = dMatProp[mat][3];
     * pmin = dMatProp[mat][4];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float mu;
    float p, ph;
//    float c;

    mu = (rho - hMatProp[mat][0]) / hMatProp[mat][0];

    ph = (hMatProp[mat][0] * powf(hMatProp[mat][1], 2) * mu*(1.0 +mu))
         / powf((1.0 - (hMatProp[mat][3] -1.0) * mu), 2);

    p = ph + hMatProp[mat][2] * hMatProp[mat][0]
        * (u - (0.5 * ph / hMatProp[mat][0] * (mu / (1.0 + mu))));

    if (p < hMatProp[mat][4]) p = hMatProp[mat][4];

//    c = hMatProp[mat][1];

    return p;
}


float pressureTaitHost(int mat, float rho, float u) {
    /**
     * \brief Tait Equation Of State
     *
     * p = rho0 * c0 * c0 / 7.0 * (powf((rho / rho0), 7) - 1.0);
     * c = c0;
     *
     * rho0 = dMatProp[mat][0];
     * c0 = dMatProp[mat][1];
     * pmin = dMatProp[mat][2];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float p;
//    float c;

    p = hMatProp[mat][0] * powf(hMatProp[mat][1], 2) / 7.0
        * (powf((rho / hMatProp[mat][0]), 7) - 1.0);

    if (p < hMatProp[mat][2]) p = hMatProp[mat][2];

//    c = hMatProp[mat][1];

    return p;
}


// Global code

void balanceMassMomentumHost(const int pn, const int* List,
		const int* Material, const float* Mass, const float* Smooth,
		const float* PosX, const float* PosY, const float* PosZ,
		const float* VelX, const float* VelY, const float* VelZ,
		const float* Density, const float* Pressure, const float* Sound,
		float* DensityDot, float* VelDotX, float* VelDotY, float* VelDotZ) {

    /**
     * \brief Interate particles
     *
     * \date Jan 6, 2011
     * \author Luca Massidda
     */

    int ip, il, jp;
    float iDensityDot;
    float iVelDotX, iVelDotY, iVelDotZ;
    float iSmooth, jMass;
    float dx, dy, dz, dr, dvr, dwdr, f, w, w0;

    for (ip = 0; ip < pn; ip++) {
        iDensityDot = 0.0f;
        iVelDotX = 0.0f;
        iVelDotY = 0.0f;
        iVelDotZ = 0.0f;
        iSmooth = Smooth[ip];
		
        for (il = 0; il < MAXN; il++) {
            jp = List[ip * MAXN + il];
            
            jMass = Mass[jp];

            dx = PosX[ip] - PosX[jp];
            dy = PosY[ip] - PosY[jp];
			dz = PosZ[ip] - PosZ[jp];
            dr = sqrtf(dx * dx + dy * dy + dz * dz);

            if (dr < (0.01f * iSmooth)) dr = 100.0f * iSmooth;

			w = kernelWendland(dr, iSmooth);
			w0 = kernelWendland(0.0f, iSmooth);
            dwdr = kernelDerivWendland(dr, iSmooth);

			dvr = 0.0f;
			dvr += (PosX[ip] - PosX[jp]) * (VelX[ip] - VelX[jp]);
			dvr += (PosY[ip] - PosY[jp]) * (VelY[ip] - VelY[jp]);
			dvr += (PosZ[ip] - PosZ[jp]) * (VelZ[ip] - VelZ[jp]);
			
			iDensityDot += jMass * dvr * dwdr / dr;
			
			// Calculate interparticle pressure action
			//f = -(Pressure[ip] + Pressure[jp])
			//	/ (Density[ip] * Density[jp]);
			f = -(Pressure[ip] / powf(Density[ip], 2) + Pressure[jp] / powf(Density[jp], 2));

			iVelDotX += jMass * f * dwdr * (PosX[ip] - PosX[jp]) / dr;
			iVelDotY += jMass * f * dwdr * (PosY[ip] - PosY[jp]) / dr;
			iVelDotZ += jMass * f * dwdr * (PosZ[ip] - PosZ[jp]) / dr;
	
			// Calculate shock correction for mass
			f = Density[ip] - Density[jp];
			f *= 2.0f * Sound[ip] / (Density[ip] + Density[jp]);

			iDensityDot += jMass * f * dwdr;

			// Calculate shock correction for momentum
			if (dvr < 0.0f) f = dvr;
            else f = 0.0f;

            f *= iSmooth / (dr * dr + 0.01f * iSmooth * iSmooth);
            f *= 2.0f * Sound[ip] / (Density[ip] + Density[jp]);
            f *= 0.03f;

            iVelDotX += jMass * f * dwdr * (PosX[ip] - PosX[jp]) / dr;
            iVelDotY += jMass * f * dwdr * (PosY[ip] - PosY[jp]) / dr;
            iVelDotZ += jMass * f * dwdr * (PosZ[ip] - PosZ[jp]) / dr;
			
			// Calculate boundary repulsion
            if (Material[ip] != Material[jp]) {
				f = 0.02f * w / w0 * Sound[ip] * Sound[jp] / iSmooth;
				iVelDotX += jMass  / (Mass[ip] + jMass) * f * (PosX[ip] - PosX[jp]) / dr;
				iVelDotY += jMass  / (Mass[ip] + jMass) * f * (PosY[ip] - PosY[jp]) / dr;
				iVelDotZ += jMass  / (Mass[ip] + jMass) * f * (PosZ[ip] - PosZ[jp]) / dr;
			}
        }

        DensityDot[ip] += iDensityDot;
        VelDotX[ip] += iVelDotX;
        VelDotY[ip] += iVelDotY;
        VelDotZ[ip] += iVelDotZ;
    }
}

__global__ void balanceMassMomentumDevice(const int pn, const int* List,
        const int* Material, const float* Mass, const float* Smooth,
        const float* PosX, const float* PosY, const float* PosZ,
        const float* VelX, const float* VelY, const float* VelZ,
        const float* Density, const float* Pressure, const float* Sound,
        float* DensityDot, float* VelDotX, float* VelDotY, float* VelDotZ) {

    /**
     * \brief Interate particles
     *
     * \date Jan 6, 2011
     * \author Luca Massidda
     */

    int ip, il, jp;
    float iDensityDot;
    float iVelDotX, iVelDotY, iVelDotZ;
    float iSmooth, jMass;
    volatile float dx, dy, dz, dr, dvr, dwdr, f, w, w0, q;

    ip = threadIdx.x + blockDim.x * blockIdx.x;
	
    if (ip < pn) {
        iDensityDot = 0.0f;
        iVelDotX = 0.0f;
        iVelDotY = 0.0f;
        iVelDotZ = 0.0f;
        iSmooth = Smooth[ip];

        for (il = 0; il < MAXN; il++) {
            jp = List[ip * MAXN + il];

            jMass = Mass[jp];

            dx = PosX[ip] - PosX[jp];
            dy = PosY[ip] - PosY[jp];
            dz = PosZ[ip] - PosZ[jp];
            dr = sqrtf(dx * dx + dy * dy + dz * dz);

            if (dr < (0.01f * iSmooth)) dr = 100.0f * iSmooth;
			
			w = kernelWendland(dr, iSmooth);
			dwdr = kernelDerivWendland(dr, iSmooth);
			
            if (Material[ip] == Material[jp]) {
				dvr = 0.0f;
				dvr += (PosX[ip] - PosX[jp]) * (VelX[ip] - VelX[jp]);
				dvr += (PosY[ip] - PosY[jp]) * (VelY[ip] - VelY[jp]);
				dvr += (PosZ[ip] - PosZ[jp]) * (VelZ[ip] - VelZ[jp]);
				
				iDensityDot += jMass * dvr * dwdr / dr;
				
				// Calculate interparticle pressure action
				f = -(Pressure[ip] / powf(Density[ip], 2) + Pressure[jp] / powf(Density[jp], 2));
				f *= jMass * dwdr;
				iVelDotX += f * (PosX[ip] - PosX[jp]) / dr;
				iVelDotY += f * (PosY[ip] - PosY[jp]) / dr;
				iVelDotZ += f * (PosZ[ip] - PosZ[jp]) / dr;
				
				// Calculate shock correction for mass
				f = Density[ip] - Density[jp];
				f *= 2.0f * Sound[ip] / (Density[ip] + Density[jp]);
				iDensityDot += jMass * f * dwdr;
				
				// Calculate shock correction for momentum
				if (dvr < 0.0f) f = dvr;
				else f = 0.0f;
				
				f *= iSmooth / (dr * dr + 0.01f * iSmooth * iSmooth);
				f *= 2.0f * Sound[ip] / (Density[ip] + Density[jp]);
				f *= 0.03f;
				f *= jMass * dwdr;
				
				iVelDotX += f * (PosX[ip] - PosX[jp]) / dr;
				iVelDotY += f * (PosY[ip] - PosY[jp]) / dr;
				iVelDotZ += f * (PosZ[ip] - PosZ[jp]) / dr;
			}
			
			// Calculate boundary repulsion
            if (Material[ip] != Material[jp]) {
				f = 0.25f * w * Mass[jp] / Density[jp] / Smooth[jp] * powf(Sound[jp], 2);
				iVelDotX += f * (PosX[ip] - PosX[jp]) / dr;
				iVelDotY += f * (PosY[ip] - PosY[jp]) / dr;
				iVelDotZ += f * (PosZ[ip] - PosZ[jp]) / dr;
			}
        }
		
        DensityDot[ip] += iDensityDot;
        VelDotX[ip] += iVelDotX;
        VelDotY[ip] += iVelDotY;
        VelDotZ[ip] += iVelDotZ;
    }
}

void balanceMassMomentumHostOld(void) {
	
    /**
     * \brief Interate particles
     *
     * \date Jan 6, 2011
     * \author Luca Massidda
     */

	int ip, il, jp;
    float iDensityDot;
    float iVelDotX, iVelDotY;
    float dx, dy, dz, dr, dvr, dwdr, f;
	
    for (ip = 0; ip < hPN; ip++) {
		iDensityDot = 0.0;
		iVelDotX = 0.0;
		iVelDotY = 0.0;
		
		for (il = 0; il < MAXN; il++) {
			jp = hList[ip * MAXN + il];
			
			dx = hPosX[ip] - hPosX[jp];
			dy = hPosY[ip] - hPosY[jp];
			dz = 0.0;
			dr = sqrtf(dx * dx + dy * dy + dz * dz);
			
			if (dr < 0.1 * hSmooth) dr = 100.0 * hSmooth;
			
			//dwdr = kernelDerivGauss(dr, hSmooth);
			dwdr = kernelDerivWendland(dr, hSmooth);
			
			dvr = 0.0;
			dvr += (hPosX[ip] - hPosX[jp]) * (hVelX[ip] - hVelX[jp]);
			dvr += (hPosY[ip] - hPosY[jp]) * (hVelY[ip] - hVelY[jp]);
			
			iDensityDot += hMass * dvr * dwdr / dr;
			
			// Calculate interparticle pressure action
			f = -(hPressure[ip] + hPressure[jp])
				/ (hDensity[ip] * hDensity[jp]);
			
			iVelDotX += hMass * f * dwdr * (hPosX[ip] - hPosX[jp]) / dr;
			iVelDotY += hMass * f * dwdr * (hPosY[ip] - hPosY[jp]) / dr;
			
			// Calculate shock correction for mass
			f = hDensity[ip] - hDensity[jp];
			f *= 2.0 * hSound / (hDensity[ip] + hDensity[jp]);
			
			iDensityDot += hMass * f * dwdr;
			
			// Calculate shock correction for momentum
			if (dvr < 0) f = dvr;
			else f = 0.0;
			
			f *= hSmooth / (dr * dr + 0.01 * hSmooth * hSmooth);
			f *= 2. * hSound / (hDensity[ip] + hDensity[jp]);
			f *= 0.03;
			
			iVelDotX += hMass * f * dwdr * (hPosX[ip] - hPosX[jp]) / dr;
			iVelDotY += hMass * f * dwdr * (hPosY[ip] - hPosY[jp]) / dr;
		}
		
		hDensityDot[ip] += iDensityDot;
		hVelDotX[ip] += iVelDotX;
		hVelDotY[ip] += iVelDotY;

    }
}


void balanceEnergyHost(const int pn,
                                const float* Pressure, const float* Density,
                                const float* DensityDot, float* EnergyDot) {

    /**
     * \brief Interate particles
     *
     * \date Jan 9, 2011
     * \author Luca Massidda
     */

    int ip;
    float iPressure, iDensity, iDensityDot;
    float iEnergyDot;

    for (ip = 0; ip < pn; ip++) {
        iPressure = Pressure[ip];
        iDensity = Density[ip];
        iDensityDot = DensityDot[ip];

        iEnergyDot = (iPressure * iDensityDot) / (iDensity * iDensity);

        EnergyDot[ip] += iEnergyDot;
    }
}

__global__ void balanceEnergyDevice(const int pn,
                                    const float* Pressure, const float* Density,
                                    const float* DensityDot, float* EnergyDot) {

    /**
     * \brief Interate particles
     *
     * \date Jan 9, 2011
     * \author Luca Massidda
     */

    volatile int ip;
    float iPressure, iDensity, iDensityDot;
    float iEnergyDot;

    ip = threadIdx.x + blockDim.x * blockIdx.x;

    if (ip < pn) {
        iPressure = Pressure[ip];
        iDensity = Density[ip];
        iDensityDot = DensityDot[ip];

        iEnergyDot = (iPressure * iDensityDot) / (iDensity * iDensity);

        EnergyDot[ip] += iEnergyDot;
    }
}

void balanceEnergyHostOld(void) {

    /**
     * \brief Interate particles
     *
     * \date Jan 9, 2011
     * \author Luca Massidda
     */

    int ip;
    float iPressure, iDensity, iDensityDot;
    float iEnergyDot;

    for (ip = 0; ip < hPN; ip++) {
        iPressure = hPressure[ip];
        iDensity = hDensity[ip];
        iDensityDot = hDensityDot[ip];

        iEnergyDot = (iPressure * iDensityDot) / (iDensity * iDensity);

        hEnergyDot[ip] += iEnergyDot;
    }
}



__host__ __device__ float pressureGas(float* properties, float rho, float u) {
    /**
     * \brief Ideal gas Equation Of State
     *
     * p = (k -1) rho u
     * c = (k(k -1) u)^0.5
     *
     * k = properties[1]
     * pshift = properties[2]
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float p;

    p = (properties[1] - 1.0f) * rho * u;
    p += properties[2];

    return p;
}



__host__ __device__ float pressurePoly(float* properties, float rho, float u) {
    /**
     * \brief Mie-Gruneisen polynomial Equation Of State
     *
     * p = a1 mu + a2 mu^2 + a3 mu^3 + (b0 + b1 mu) rho0 u  in compression
     * p = t1 mu + t2 mu^2 + b0 rho0 u                      in tension
     *
     * rho0 = properties[0];
     * a1 = properties[1];
     * a2 = properties[2];
     * a3 = properties[3];
     * b0 = properties[4];
     * b1 = properties[5];
     * t1 = properties[6];
     * t2 = properties[7];
     * pmin = properties[8];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float mu;
    float p;

    mu = (rho - properties[0]) / properties[0];

    if (mu < 0)
        p = (properties[6] * mu + properties[7] * mu*mu)
            + (properties[4] * properties[0] * u);
    else
        p = (properties[1] * mu + properties[2] * mu*mu
             + properties[3] * mu*mu*mu)
            + ((properties[4] + properties[5] * mu)
               * properties[0] * u);

    //if (p < properties[8]) p = properties[8];

    return p;
}

__host__ __device__ float pressureShock(float* properties, float rho, float u) {
    /**
     * \brief Mie-Gruneisen Shock Hugoniot Equation Of State
     *
     * mu = rho / rho0 -1
     * g = g * rho0 / rho
     * ph = (rho0 c0^2 mu (1 + mu)) / (1 - (s0 - 1) * mu)^2
     * uh = 1/2 ph/rho0 * (mu / (1 + mu))
     * p = ph + g * rho * (u - uh)
     *
     * rho0 = properties[0];
     * c0 = properties[1];
     * g0 = properties[2];
     * s0 = properties[3];
     * pmin = properties[4];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float mu;
    float p, ph;

    mu = (rho - properties[0]) / properties[0];

    ph = (properties[0] * powf(properties[1], 2) * mu*(1.0f +mu))
         / powf((1.0f - (properties[3] -1.0f) * mu), 2);

    p = ph + properties[2] * properties[0]
        * (u - (0.5f * ph / properties[0] * (mu / (1.0f + mu))));

    //if (p < properties[4]) p = properties[4];

    return p;
}


__host__ __device__ float pressureTait(float* properties, float rho, float u) {
    /**
     * \brief Tait Equation Of State
     *
     * p = rho0 * c0 * c0 / 7.0 * (powf((rho / rho0), 7) - 1.0);
     * c = c0;
     *
     * rho0 = properties[0];
     * c0 = properties[1];
     * pmin = properties[2];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float p;

    p = properties[0] * powf(properties[1], 2) / 7.0f
        * (powf((rho / properties[0]), 7) - 1.0f);

    //if (p < properties[2]) p = properties[2];

    return p;
}


__host__ __device__ float soundGas(float* properties ,float rho, float u) {
    /**
     * \brief Ideal gas Equation Of State
     *
     * p = (k -1) rho u
     * c = (k(k -1) u)^0.5
     *
     * k = properties[1]
     * pshift = properties[2]
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float c;

    c = sqrtf(properties[1] * (properties[1] - 1.0f) * u);

    return c;
}



__host__ __device__ float soundPoly(float* properties , float rho, float u) {
    /**
     * \brief Mie-Gruneisen polynomial Equation Of State
     *
     * p = a1 mu + a2 mu^2 + a3 mu^3 + (b0 + b1 mu) rho0 u  in compression
     * p = t1 mu + t2 mu^2 + b0 rho0 u                      in tension
     *
     * rho0 = properties[0];
     * a1 = properties[1];
     * a2 = properties[2];
     * a3 = properties[3];
     * b0 = properties[4];
     * b1 = properties[5];
     * t1 = properties[6];
     * t2 = properties[7];
     * pmin = properties[8];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float c;

    c = sqrtf(properties[1] / rho);

    return c;
}

__host__ __device__ float soundShock(float* properties, float rho, float u) {
    /**
     * \brief Mie-Gruneisen Shock Hugoniot Equation Of State
     *
     * mu = rho / rho0 -1
     * g = g * rho0 / rho
     * ph = (rho0 c0^2 mu (1 + mu)) / (1 - (s0 - 1) * mu)^2
     * uh = 1/2 ph/rho0 * (mu / (1 + mu))
     * p = ph + g * rho * (u - uh)
     *
     * rho0 = properties[0];
     * c0 = properties[1];
     * g0 = properties[2];
     * s0 = properties[3];
     * pmin = properties[4];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float c;

    c = properties[1];

    return c;
}


__host__ __device__ float soundTait(float* properties, float rho, float u) {
    /**
     * \brief Tait Equation Of State
     *
     * p = rho0 * c0 * c0 / 7.0 * (powf((rho / rho0), 7) - 1.0);
     * c = c0;
     *
     * rho0 = properties[0];
     * c0 = properties[1];
     * pmin = properties[2];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float c;

    c = properties[1];

    return c;
}


__host__ __device__ float densityPoly(float* properties , float rho) {
    /**
     * \brief Mie-Gruneisen polynomial Equation Of State
     *
     * p = a1 mu + a2 mu^2 + a3 mu^3 + (b0 + b1 mu) rho0 u  in compression
     * p = t1 mu + t2 mu^2 + b0 rho0 u                      in tension
     *
     * rho0 = properties[0];
     * a1 = properties[1];
     * a2 = properties[2];
     * a3 = properties[3];
     * b0 = properties[4];
     * b1 = properties[5];
     * t1 = properties[6];
     * t2 = properties[7];
     * pmin = properties[8];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float rho0;
    
    rho0 = properties[0];
    if (rho < 0.9f * rho0) rho = 0.9f*rho0;

    return rho;
}

__host__ __device__ float densityShock(float* properties, float rho) {
    /**
     * \brief Mie-Gruneisen Shock Hugoniot Equation Of State
     *
     * mu = rho / rho0 -1
     * g = g * rho0 / rho
     * ph = (rho0 c0^2 mu (1 + mu)) / (1 - (s0 - 1) * mu)^2
     * uh = 1/2 ph/rho0 * (mu / (1 + mu))
     * p = ph + g * rho * (u - uh)
     *
     * rho0 = properties[0];
     * c0 = properties[1];
     * g0 = properties[2];
     * s0 = properties[3];
     * pmin = properties[4];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float rho0;
    
    rho0 = properties[0];
    if (rho < 0.9f * rho0) rho = 0.9f*rho0;

    return rho;
}


__host__ __device__ float densityTait(float* properties, float rho) {
    /**
     * \brief Tait Equation Of State
     *
     * p = rho0 * c0 * c0 / 7.0 * (powf((rho / rho0), 7) - 1.0);
     * c = c0;
     *
     * rho0 = properties[0];
     * c0 = properties[1];
     * pmin = properties[2];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float rho0;
    
    rho0 = properties[0];
    if (rho < 0.9f * rho0) rho = 0.9f*rho0;

    return rho;
}


void updateParticlesHost(const int pn, const float alpha,
                                  const int* Material,
                                  const float* VelDotX, const float* VelDotY, const float* VelDotZ,
                                  const float* DensityDot, const float* EnergyDot,
                                  const float* PosX0, const float* PosY0, const float* PosZ0,
                                  const float* VelX0, const float* VelY0, const float* VelZ0,
                                  const float* Density0, const float* Energy0,
                                  float* PosX, float* PosY, float* PosZ,
                                  float* VelX, float* VelY, float* VelZ,
                                  float* Density, float* Energy, float* Pressure, float* Sound) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    int ip, i;
    int iMaterial;

    for (ip = 0; ip < pn; ip++) if (Material[ip] != 0) {
        PosX[ip] = PosX0[ip] + alpha * (PosX[ip] + hRun.dt * VelX[ip] - PosX0[ip]);
        PosY[ip] = PosY0[ip] + alpha * (PosY[ip] + hRun.dt * VelY[ip] - PosY0[ip]);
        PosZ[ip] = PosZ0[ip] + alpha * (PosZ[ip] + hRun.dt * VelZ[ip] - PosZ0[ip]);

        VelX[ip] = VelX0[ip] + alpha * (VelX[ip] + hRun.dt * VelDotX[ip] - VelX0[ip]);
        VelY[ip] = VelY0[ip] + alpha * (VelY[ip] + hRun.dt * VelDotY[ip] - VelY0[ip]);
        VelZ[ip] = VelZ0[ip] + alpha * (VelZ[ip] + hRun.dt * VelDotZ[ip] - VelZ0[ip]);
		//VelZ[ip] = 0.0f;

        Density[ip] = Density0[ip] + alpha * (Density[ip] + hRun.dt * DensityDot[ip] - Density0[ip]);

        Energy[ip] = Energy0[ip] + alpha * (Energy[ip] + hRun.dt * EnergyDot[ip] - Energy0[ip]);

        iMaterial = Material[ip];

        if (iMaterial <= 0) {
            VelX[ip] = VelX0[ip];
            VelY[ip] = VelY0[ip];
            VelZ[ip] = VelZ0[ip];
        }
		
		for (i = 0; i < 10; i++) 
			if ((PosX[ip] > hFix[i].minX) && 
				(PosX[ip] < hFix[i].maxX) && 
				(PosY[ip] > hFix[i].minY) && 
				(PosY[ip] < hFix[i].maxY) && 
				(PosZ[ip] > hFix[i].minZ) && 
				(PosZ[ip] < hFix[i].maxZ)) {
					VelX[ip] = hFix[i].velX;
					VelY[ip] = hFix[i].velY;
					VelZ[ip] = hFix[i].velZ;
		}
		
        iMaterial = abs(iMaterial);

        if (hMatType[iMaterial] == 0) {
            VelX[ip] = VelX0[ip];
            VelY[ip] = VelY0[ip];
            VelZ[ip] = VelZ0[ip];
            Density[ip] = Density0[ip];
            Energy[ip] = Energy0[ip];
        }
		
        switch (hMatType[iMaterial]) {
        case (0) : // BOUNDARY
            Density[ip] = densityTait(hMatProp[iMaterial], Density[ip]);
            Pressure[ip] = 0.0f*pressureTait(hMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundTait(hMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        case (1) : // IDEAL GAS EOS
            Pressure[ip] = pressureGas(hMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundGas(hMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        case (2) : // MIE-GRUNEISEN POLYNOMIAL EOS
            Density[ip] = densityPoly(hMatProp[iMaterial], Density[ip]);
            Pressure[ip] = pressurePoly(hMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundPoly(hMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        case (3) : // MIE-GRUNEISEN SHOCK EOS
            Density[ip] = densityShock(hMatProp[iMaterial], Density[ip]);
            Pressure[ip] = pressureShock(hMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundShock(hMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        case (4) : // TAIT EOS
            Density[ip] = densityTait(hMatProp[iMaterial], Density[ip]);
            Pressure[ip] = pressureTait(hMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundTait(hMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        default :
            Pressure[ip] = 0.0f;
        }

    }
}

__global__ void updateParticlesDevice(const int pn, const float alpha,
                                      const int* Material,
                                      const float* VelDotX, const float* VelDotY, const float* VelDotZ,
                                      const float* DensityDot, const float* EnergyDot,
                                      const float* PosX0, const float* PosY0, const float* PosZ0,
                                      const float* VelX0, const float* VelY0, const float* VelZ0,
                                      const float* Density0, const float* Energy0,
                                      float* PosX, float* PosY, float* PosZ,
                                      float* VelX, float* VelY, float* VelZ,
                                      float* Density, float* Energy, float* Pressure, float* Sound) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    int ip, i;
    int iMaterial;

    ip = threadIdx.x + blockDim.x * blockIdx.x;

    if (ip < pn) {
        PosX[ip] = PosX0[ip] + alpha * (PosX[ip] + dRun.dt * VelX[ip] - PosX0[ip]);
        PosY[ip] = PosY0[ip] + alpha * (PosY[ip] + dRun.dt * VelY[ip] - PosY0[ip]);
        PosZ[ip] = PosZ0[ip] + alpha * (PosZ[ip] + dRun.dt * VelZ[ip] - PosZ0[ip]);

        VelX[ip] = VelX0[ip] + alpha * (VelX[ip] + dRun.dt * VelDotX[ip] - VelX0[ip]);
        VelY[ip] = VelY0[ip] + alpha * (VelY[ip] + dRun.dt * VelDotY[ip] - VelY0[ip]);
        VelZ[ip] = VelZ0[ip] + alpha * (VelZ[ip] + dRun.dt * VelDotZ[ip] - VelZ0[ip]);
        //VelZ[ip] = 0.0f;

        Density[ip] = Density0[ip] + alpha * (Density[ip] + dRun.dt * DensityDot[ip] - Density0[ip]);

        Energy[ip] = Energy0[ip] + alpha * (Energy[ip] + dRun.dt * EnergyDot[ip] - Energy0[ip]);

        iMaterial = Material[ip];

		for (i = 0; i < 10; i++) 
			if ((PosX[ip] > dFix[i].minX) && 
				(PosX[ip] < dFix[i].maxX) && 
				(PosY[ip] > dFix[i].minY) && 
				(PosY[ip] < dFix[i].maxY) && 
				(PosZ[ip] > dFix[i].minZ) && 
				(PosZ[ip] < dFix[i].maxZ)) {
					VelX[ip] = dFix[i].velX;
					VelY[ip] = dFix[i].velY;
					VelZ[ip] = dFix[i].velZ;
		}
		
        if (dMatType[iMaterial] == 0) {
            VelX[ip] = VelX0[ip];
            VelY[ip] = VelY0[ip];
            VelZ[ip] = VelZ0[ip];
        }

        switch (dMatType[iMaterial]) {
        case (0) : // BOUNDARY
            Density[ip] = densityTait(dMatProp[iMaterial], Density[ip]);
            Pressure[ip] = 0.0f*pressureTait(dMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundTait(dMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        case (1) : // IDEAL GAS EOS
            Pressure[ip] = pressureGas(dMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundGas(dMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        case (2) : // MIE-GRUNEISEN POLYNOMIAL EOS
            Density[ip] = densityPoly(dMatProp[iMaterial], Density[ip]);
            Pressure[ip] = pressurePoly(dMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundPoly(dMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        case (3) : // MIE-GRUNEISEN SHOCK EOS
            Density[ip] = densityShock(dMatProp[iMaterial], Density[ip]);
            Pressure[ip] = pressureShock(dMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundShock(dMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        case (4) : // TAIT EOS
            Density[ip] = densityTait(dMatProp[iMaterial], Density[ip]);
            Pressure[ip] = pressureTait(dMatProp[iMaterial], Density[ip], Energy[ip]);
            Sound[ip] = soundTait(dMatProp[iMaterial], Density[ip], Energy[ip]);
            break;
        default :
            Pressure[ip] = 0.0f;
        }

    }
}


void updateParticlesHostOld(const float alpha) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    int ip;
    float f;
    int iMaterial;
    float iDensity, iEnergy;

    for (ip = 0; ip < hPN; ip++) {
        f = hPosX0[ip] + alpha * (hPosX[ip] + hRun.dt * hVelX[ip] - hPosX0[ip]);
        /*
		if (f < hRun.minX)
			f += hRun.maxX - hRun.minX;
		if (f > hRun.maxX)
			f -= hRun.maxX - hRun.minX;
		*/
        hPosX[ip] = f;

        f = hPosY0[ip] + alpha * (hPosY[ip] + hRun.dt * hVelY[ip] - hPosY0[ip]);
        /*
		if (f < hRun.minY)
			f += hRun.maxY - hRun.minY;
		if (f > hRun.maxY)
			f -= hRun.maxY - hRun.minY;
		*/
        hPosY[ip] = f;

        f = hVelX0[ip] + alpha * (hVelX[ip] + hRun.dt * hVelDotX[ip] - hVelX0[ip]);
        hVelX[ip] = f;
        
        f = hVelY0[ip] + alpha * (hVelY[ip] + hRun.dt * hVelDotY[ip] - hVelY0[ip]);
        hVelY[ip] = f;

        f = hDensity0[ip] + alpha * (hDensity[ip] + hRun.dt * hDensityDot[ip] - hDensity0[ip]);
        hDensity[ip] = f;

        f = hEnergy0[ip] + alpha * (hEnergy[ip] + hRun.dt * hEnergyDot[ip] - hEnergy0[ip]);
        hEnergy[ip] = f;
        
        iMaterial = hMaterial[ip];
        
        if (iMaterial < 0) {
			hVelX[ip] = hVelX0[ip];
			hVelY[ip] = hVelY0[ip];
        }

        iMaterial = abs(iMaterial);
        iDensity = hDensity[ip];
        iEnergy = hEnergy[ip];

        switch (hMatType[iMaterial]) {
        case (1) : // IDEAL GAS EOS
            hPressure[ip] = pressureGasHost(iMaterial, iDensity, iEnergy);
            break;
        case (2) : // MIE-GRUNEISEN POLYNOMIAL EOS
            hPressure[ip] = pressurePolyHost(iMaterial, iDensity, iEnergy);
            break;
        case (3) : // MIE-GRUNEISEN SHOCK EOS
            hPressure[ip] = pressureShockHost(iMaterial, iDensity, iEnergy);
            break;
        case (4) : // TAIT EOS
            hPressure[ip] = pressureTaitHost(iMaterial, iDensity, iEnergy);
            break;
        default :
            hPressure[ip] = 0.0;
        }
        
        
	}
}


void updateLoadsHost(const int pn,
                              const int* Material,
                              float* PosX, float* PosY, float* PosZ,
                              float* VelX, float* VelY, float* VelZ,
                              float* VelDotX, float* VelDotY, float* VelDotZ,
                              float* EnergyDot) {

    int ip, i;

    for (ip = 0; ip < pn; ip++) {
		if (Material[ip] > 0) {
			for (i = 0; i < 10; i++) {
				if ((PosX[ip] > hLoad[i].minX) &&
					(PosX[ip] < hLoad[i].maxX) &&
                    (PosZ[ip] < hLoad[i].maxZ) &&
                    (PosY[ip] > hLoad[i].minY) &&
                    (PosY[ip] < hLoad[i].maxY) &&
                    (PosZ[ip] > hLoad[i].minZ) &&
                    (PosZ[ip] < hLoad[i].maxZ)) {
					VelDotX[ip] += hLoad[i].gx;
					VelDotY[ip] += hLoad[i].gy;
					VelDotZ[ip] += hLoad[i].gz;
					EnergyDot[ip] += hLoad[i].w;
				}
            }
        }

    }

}

__global__ void updateLoadsDevice(const int pn, const int* Material, 
                                  const float* PosX, const float* PosY, const float* PosZ,
                                  float* VelDotX, float* VelDotY, float* VelDotZ,
                                  float* EnergyDot) {

    int ip, i;

    ip = threadIdx.x + blockDim.x * blockIdx.x;

    if ((ip < pn) && (Material[ip] > 0)) {
        for (i = 0; i < 10; i++) {
            if ((PosX[ip] > dLoad[i].minX) &&
                    (PosX[ip] < dLoad[i].maxX) &&
                    (PosZ[ip] < dLoad[i].maxZ) &&
                    (PosY[ip] > dLoad[i].minY) &&
                    (PosY[ip] < dLoad[i].maxY) &&
                    (PosZ[ip] > dLoad[i].minZ) &&
                    (PosZ[ip] < dLoad[i].maxZ)) {
                VelDotX[ip] += dLoad[i].gx;
                VelDotY[ip] += dLoad[i].gy;
                VelDotZ[ip] += dLoad[i].gz;
                EnergyDot[ip] += dLoad[i].w;
            }
        }
    }

}

void updateForcesHost(void) {
	
    int ip;
    int iMaterial;
    float iVelDotX, iVelDotY, iDensityDot, iEnergyDot;

    for (ip = 0; ip < hPN; ip++) {
        iVelDotX = 0.0;
        iVelDotY = 0.0;
        iDensityDot = 0.0;
        iEnergyDot = 0.0;

        iMaterial = hMaterial[ip];

        if (iMaterial > 0) iVelDotY = -9.81;

        hVelDotX[ip] = iVelDotX;
        hVelDotY[ip] = iVelDotY;
        hDensityDot[ip] = iDensityDot;
        hEnergyDot[ip] = iEnergyDot;
    }
}



// Host code

int initHost(struct model *hm) {

    hm->Material = (int *) malloc(MAXP * sizeof(int));
    hm->Mass = (float *) malloc(MAXP * sizeof(float));
    hm->Smooth = (float *) malloc(MAXP * sizeof(float));
    hm->PosX = (float *) malloc(MAXP * sizeof(float));
    hm->PosY = (float *) malloc(MAXP * sizeof(float));
    hm->PosZ = (float *) malloc(MAXP * sizeof(float));
    hm->VelX = (float *) malloc(MAXP * sizeof(float));
    hm->VelY = (float *) malloc(MAXP * sizeof(float));
    hm->VelZ = (float *) malloc(MAXP * sizeof(float));
    hm->Density = (float *) malloc(MAXP * sizeof(float));
    hm->Energy = (float *) malloc(MAXP * sizeof(float));
    hm->Pressure = (float *) malloc(MAXP * sizeof(float));
    hm->Sound = (float *) malloc(MAXP * sizeof(float));
    hm->VelDotX = (float *) malloc(MAXP * sizeof(float));
    hm->VelDotY = (float *) malloc(MAXP * sizeof(float));
    hm->VelDotZ = (float *) malloc(MAXP * sizeof(float));
    hm->DensityDot = (float *) malloc(MAXP * sizeof(float));
    hm->EnergyDot = (float *) malloc(MAXP * sizeof(float));
    hm->PosX0 = (float *) malloc(MAXP * sizeof(float));
    hm->PosY0 = (float *) malloc(MAXP * sizeof(float));
    hm->PosZ0 = (float *) malloc(MAXP * sizeof(float));
    hm->VelX0 = (float *) malloc(MAXP * sizeof(float));
    hm->VelY0 = (float *) malloc(MAXP * sizeof(float));
    hm->VelZ0 = (float *) malloc(MAXP * sizeof(float));
    hm->Density0 = (float *) malloc(MAXP * sizeof(float));
    hm->Energy0 = (float *) malloc(MAXP * sizeof(float));
    
    hm->Hash = (int *) malloc(MAXP * sizeof(int));
    hm->Index = (int *) malloc(MAXP * sizeof(int));
    hm->List = (int *) malloc(MAXP * MAXN * sizeof(int));
    hm->IntDummy = (int *) malloc(MAXP * sizeof(int));
    hm->FloatDummy = (float *) malloc(MAXP * sizeof(float));
	
    hm->SetStart = (int *) malloc(MAXG * sizeof(int));
    hm->SetStop = (int *) malloc(MAXG * sizeof(int));
    
    
	hMaterial = hm->Material;
	hPosX = hm->PosX;
	hPosY = hm->PosY;
    hVelX = hm->VelX;
    hVelY = hm->VelY;
    hDensity = hm->Density;
    hEnergy = hm->Energy;
    hPressure = hm->Pressure;
    hVelDotX = hm->VelDotX;
    hVelDotY = hm->VelDotY;
    hDensityDot = hm->DensityDot;
    hEnergyDot = hm->EnergyDot;
    hPosX0 = hm->PosX0;
    hPosY0 = hm->PosY0;
    hVelX0 = hm->VelX0;
    hVelY0 = hm->VelY0;
    hDensity0 = hm->Density0;
    hEnergy0 = hm->Energy0;
	
    hList = hm->List;
    hHash = hm->Hash;
    hIndex = hm->Index;
    hSetStart = hm->SetStart;
    hSetStop = hm->SetStop;
	
    return 0;
}

int initDevice(struct model *dm) {
	size_t available, total;
	
    hipMalloc((void**) &(dm->Material), (MAXP * sizeof(int)));
    hipMalloc((void**) &(dm->Mass), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->Smooth), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->PosX), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->PosY), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->PosZ), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelX), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelY), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelZ), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->Density), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->Energy), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->Pressure), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->Sound), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelDotX), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelDotY), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelDotZ), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->DensityDot), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->EnergyDot), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->PosX0), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->PosY0), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->PosZ0), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelX0), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelY0), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->VelZ0), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->Density0), (MAXP * sizeof(float)));
    hipMalloc((void**) &(dm->Energy0), (MAXP * sizeof(float)));
    
    hipMalloc((void**) &(dm->Hash), (MAXP * sizeof(int)));
    hipMalloc((void**) &(dm->Index), (MAXP * sizeof(int)));
    hipMalloc((void**) &(dm->List), (MAXP * MAXN * sizeof(int)));
    hipMalloc((void**) &(dm->IntDummy), (MAXP * sizeof(int)));
    hipMalloc((void**) &(dm->FloatDummy), (MAXP * sizeof(float)));
	
    hipMalloc((void**) &(dm->SetStart), (MAXG * sizeof(int)));
    hipMalloc((void**) &(dm->SetStop), (MAXG * sizeof(int)));
    
	hipMemGetInfo(&available, &total);
	printf("Available memory %d of %d MB\n", available/1024/1024, total/1024/1024);
	
    return 0;
}

int copyHostToDevice(struct model *hm, struct model *dm) {

    dm->pn = hm->pn;
    
	hipMemcpy(dm->Material, hm->Material, (MAXP * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Mass, hm->Mass, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Smooth, hm->Smooth, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->PosX, hm->PosX, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->PosY, hm->PosY, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->PosZ, hm->PosZ, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelX, hm->VelX, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelY, hm->VelY, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelZ, hm->VelZ, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Density, hm->Density, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Energy, hm->Energy, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Pressure, hm->Pressure, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Sound, hm->Sound, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelDotX, hm->VelDotX, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelDotY, hm->VelDotY, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelDotZ, hm->VelDotZ, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->DensityDot, hm->DensityDot, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->EnergyDot, hm->EnergyDot, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->PosX0, hm->PosX0, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->PosY0, hm->PosY0, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->PosZ0, hm->PosZ0, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelX0, hm->VelX0, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelY0, hm->VelY0, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->VelZ0, hm->VelZ0, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Density0, hm->Density0, (MAXP * sizeof(float)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Energy0, hm->Energy0, (MAXP * sizeof(float)), hipMemcpyHostToDevice);

    hipMemcpy(dm->List, hm->List, (MAXP * MAXN * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Hash, hm->Hash, (MAXP * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dm->Index, hm->Index, (MAXP * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dm->IntDummy, hm->IntDummy, (MAXP * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dm->FloatDummy, hm->FloatDummy, (MAXP * sizeof(float)), hipMemcpyHostToDevice);

    hipMemcpy(dm->SetStart, hm->SetStart, (MAXG * sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpy(dm->SetStop, hm->SetStop, (MAXG * sizeof(int)), hipMemcpyHostToDevice);

    dGrid.oX = hGrid.oX;
    dGrid.oY = hGrid.oY;
    dGrid.oZ = hGrid.oZ;
    dGrid.nX = hGrid.nX;
    dGrid.nY = hGrid.nY;
    dGrid.nZ = hGrid.nZ;
    dGrid.size = hGrid.size;
    
    for (int i = 0; i < 10; i++) dIn[i] = hIn[i];

    hipMemcpyToSymbol(HIP_SYMBOL("dMatType"), hMatType, 10 * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL("dMatProp"), hMatProp, 100 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL("dRun"), &hRun, sizeof(struct simulation));
    hipMemcpyToSymbol(HIP_SYMBOL("dLoad"), &hLoad, 10 * sizeof(struct load));
    hipMemcpyToSymbol(HIP_SYMBOL("dFix"), &hFix, 10 * sizeof(struct fix));
    hipMemcpyToSymbol(HIP_SYMBOL("dOut"), &hOut, 10 * sizeof(struct outlet));
    
    return 0;
}

int copyDeviceToHost(struct model *dm, struct model *hm) {

    hm->pn = dm->pn;
    
	hipMemcpy(hm->Material, dm->Material, (MAXP * sizeof(int)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Mass, dm->Mass, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Smooth, dm->Smooth, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->PosX, dm->PosX, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->PosY, dm->PosY, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->PosZ, dm->PosZ, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelX, dm->VelX, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelY, dm->VelY, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelZ, dm->VelZ, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Density, dm->Density, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Energy, dm->Energy, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Pressure, dm->Pressure, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Sound, dm->Sound, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelDotX, dm->VelDotX, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelDotY, dm->VelDotY, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelDotZ, dm->VelDotZ, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->DensityDot, dm->DensityDot, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->EnergyDot, dm->EnergyDot, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->PosX0, dm->PosX0, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->PosY0, dm->PosY0, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->PosZ0, dm->PosZ0, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelX0, dm->VelX0, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelY0, dm->VelY0, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->VelZ0, dm->VelZ0, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Density0, dm->Density0, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Energy0, dm->Energy0, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);

    hipMemcpy(hm->List, dm->List, (MAXP * MAXN * sizeof(int)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Hash, dm->Hash, (MAXP * sizeof(int)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->Index, dm->Index, (MAXP * sizeof(int)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->IntDummy, dm->IntDummy, (MAXP * sizeof(int)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->FloatDummy, dm->FloatDummy, (MAXP * sizeof(float)), hipMemcpyDeviceToHost);

    hipMemcpy(hm->SetStart, dm->SetStart, (MAXG * sizeof(int)), hipMemcpyDeviceToHost);
    hipMemcpy(hm->SetStop, dm->SetStop, (MAXG * sizeof(int)), hipMemcpyDeviceToHost);

    hGrid.oX = dGrid.oX;
    hGrid.oY = dGrid.oY;
    hGrid.oZ = dGrid.oZ;
    hGrid.nX = dGrid.nX;
    hGrid.nY = dGrid.nY;
    hGrid.nZ = dGrid.nZ;
    hGrid.size = dGrid.size;
    
    for (int i = 0; i < 10; i++) hIn[i] = dIn[i];

    return 0;
}


int backupDataHost(struct model *hm) {

    memcpy(hm->PosX0, hm->PosX, MAXP * sizeof(float));
    memcpy(hm->PosY0, hm->PosY, MAXP * sizeof(float));
    memcpy(hm->PosZ0, hm->PosZ, MAXP * sizeof(float));
    memcpy(hm->VelX0, hm->VelX, MAXP * sizeof(float));
    memcpy(hm->VelY0, hm->VelY, MAXP * sizeof(float));
    memcpy(hm->VelZ0, hm->VelZ, MAXP * sizeof(float));
    memcpy(hm->Density0, hm->Density, MAXP * sizeof(float));
    memcpy(hm->Energy0, hm->Energy, MAXP * sizeof(float));

    return 0;
}

int backupDataDevice(struct model *dm) {

    hipMemcpy(dm->PosX0, dm->PosX, (MAXP * sizeof(float)), hipMemcpyDeviceToDevice);
    hipMemcpy(dm->PosY0, dm->PosY, (MAXP * sizeof(float)), hipMemcpyDeviceToDevice);
    hipMemcpy(dm->PosZ0, dm->PosZ, (MAXP * sizeof(float)), hipMemcpyDeviceToDevice);
    hipMemcpy(dm->VelX0, dm->VelX, (MAXP * sizeof(float)), hipMemcpyDeviceToDevice);
    hipMemcpy(dm->VelY0, dm->VelY, (MAXP * sizeof(float)), hipMemcpyDeviceToDevice);
    hipMemcpy(dm->VelZ0, dm->VelZ, (MAXP * sizeof(float)), hipMemcpyDeviceToDevice);
    hipMemcpy(dm->Density0, dm->Density, (MAXP * sizeof(float)), hipMemcpyDeviceToDevice);
    hipMemcpy(dm->Energy0, dm->Energy, (MAXP * sizeof(float)), hipMemcpyDeviceToDevice);
    
    return 0;
}


int backupDataHostOld() {

    memcpy(hPosX0, hPosX, MAXP * sizeof(float));
    memcpy(hPosY0, hPosY, MAXP * sizeof(float));
    memcpy(hVelX0, hVelX, MAXP * sizeof(float));
    memcpy(hVelY0, hVelY, MAXP * sizeof(float));
    memcpy(hDensity0, hDensity, MAXP * sizeof(float));
    memcpy(hEnergy0, hEnergy, MAXP * sizeof(float));

    return 0;
}


int initRun() {

    /**
     * \brief Input run data
     *
     * Reads the input file for run data
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    char tok[10];
    int i, m, p, pn;
    int iv;
    float fv;
    int mpn, mpp[10];

    // Open stream file
    stream = fopen("armando.run", "r");

    while (!feof(stream)) {
        sprintf(tok, " ");
        fscanf(stream, "%s", tok);

        if (strcmp(tok, "MAT") == 0) {
            fscanf(stream, "%i", &iv);
            if ((iv > 0) && (iv <= 50))
                m = iv;

            for (p = 0; p < 10; p++)
                hMatProp[m][p] = 0.0;

            if ((m > 0) && (m <= 10))
                pn = 3;
            if ((m > 10) && (m <= 20))
                pn = 9;
            if ((m > 20) && (m <= 30))
                pn = 10;
            if ((m > 30) && (m <= 40))
                pn = 5;
            if ((m > 40) && (m <= 50))
                pn = 3;

            for (p = 0; p < pn; p++) {
                fscanf(stream, "%f", &fv);
                hMatProp[m][p] = fv;
            }

            printf("Material %d\n", m);
            printf("hMatProp: \n");
            for (p = 0; p < pn; p++)
                printf(" %f\n", hMatProp[m][p]);
            printf("\n");
        }

        if (strcmp(tok, "TIME") == 0) {
            fscanf(stream, "%f", &fv);
            if (fv > 0.0)
                hRun.dt = fv;

            fscanf(stream, "%i", &iv);
            if (iv > 0)
                hRun.tsn = iv;

            fscanf(stream, "%i", &iv);
            if (iv > 0)
                hRun.ssi = iv;

            printf("Time step: %f\n", hRun.dt);
            printf("Steps: %i\n", hRun.tsn);
            printf("Save step: %i\n", hRun.ssi);
            printf("\n");
        }

        if (strcmp(tok, "LIMITS") == 0) {
            fscanf(stream, "%f", &fv);
            hRun.minX = fv;

            fscanf(stream, "%f", &fv);
            hRun.maxX = fv;

            fscanf(stream, "%f", &fv);
            hRun.minY = fv;

            fscanf(stream, "%f", &fv);
            hRun.maxY = fv;

            printf("Domain limits: \n");
            printf("X: %+e - %+e \n", hRun.minX, hRun.maxX);
            printf("Y: %+e - %+e \n", hRun.minY, hRun.maxY);
            printf("\n");
        }

        if (strcmp(tok, "MONITORS") == 0) {
            fscanf(stream, "%i", &iv);
            mpn = iv;

            for (i = 0; i < mpn; i++) {
                fscanf(stream, "%i", &iv);
                mpp[i] = iv;
            }

            printf("Monitored particles: %i \n", mpn);
            if (mpn > 0) {
                printf("Index:");
                for (i = 0; i < mpn; i++)
                    printf(" %i", mpp[i]);
                printf("\n");
                printf("\n");
            }
        }
    }

    fclose(stream);

    hSound = hSmooth / hRun.dt;

    return 0;
}

int scanData() {
    /**
     * \brief Input particle data file
     *
     * Reads particle data from a disk file
     *
     * \date Oct 20, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    int i;
    float fv1, fv2, fv3;
    int iv;

    // Stream file position
    stream = fopen("in_pos.txt", "r");
    for (i = 0; !feof(stream); i++) {
        fscanf(stream, "%e %e ", &fv1, &fv2);
        hPosX[i] = fv1;
        hPosY[i] = fv2;
    }
    fclose(stream);
    hPN = i;

    // Stream file velocity
    stream = fopen("in_vel.txt", "r");
    for (i = 0; i < hPN; i++) {
        fscanf(stream, "%e %e", &fv1, &fv2);
        hVelX[i] = fv1;
        hVelY[i] = fv2;
    }
    fclose(stream);

    // Stream file info
    stream = fopen("in_info.txt", "r");
    for (i = 0; i < hPN; i++) {
        fscanf(stream, "%i %e %e ", &iv, &fv1, &fv2);
        hMaterial[i] = iv;
        hMass = fv1;
        hSmooth = fv2;
    }
    fclose(stream);

    // Stream file field
    stream = fopen("in_field.txt", "r");
    for (i = 0; i < hPN; i++) {
        fscanf(stream, "%e %e %e ", &fv1, &fv2, &fv3);
        hDensity[i] = fv1;
        hPressure[i] = fv2;
        hEnergy[i] = fv3;
    }
    fclose(stream);

    return 0;
}

int printDataOld() {
    /**
     * \brief Particle data file output
     *
     * Saves particle data on a disk file
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    int i;

    // Stream file position
    stream = fopen("new_pos.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e\n", hPosX[i], hPosY[i]);
    fclose(stream);

    // Stream file velocity
    stream = fopen("new_vel.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e \n", hVelX[i], hVelY[i]);
    fclose(stream);

    // Stream file info
    stream = fopen("new_info.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%i %+14.8e %+14.8e \n", hMaterial[i], hMass, hSmooth);
    fclose(stream);

    // Stream file field
    stream = fopen("new_field.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e \n", hDensity[i], hPressure[i], hEnergy[i]);
    fclose(stream);

    // Stream file add1
    stream = fopen("new_debug.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e %+14.8e \n", hDensityDot[i],
                hVelDotX[i], hVelDotY[i], hEnergyDot[i]);
    fclose(stream);

    return 0;
}

int printData(struct model *hm) {
    /**
     * \brief Particle data file output
     *
     * Saves particle data on a disk file
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    int i;

    // Stream file position
    stream = fopen("new_pos.txt", "w");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e \n", hm->PosX[i], hm->PosY[i], hm->PosZ[i]);
    fclose(stream);

    // Stream file velocity
    stream = fopen("new_vel.txt", "w");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e \n", hm->VelX[i], hm->VelY[i], hm->VelZ[i]);
    fclose(stream);

    // Stream file info
    stream = fopen("new_info.txt", "w");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%i %+14.8e %+14.8e \n", hm->Material[i], hm->Mass[i], hm->Smooth[i]);
    fclose(stream);

    // Stream file field
    stream = fopen("new_field.txt", "w");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e \n", hm->Density[i], hm->Pressure[i], hm->Energy[i]);
    fclose(stream);
	/*
    // Stream file add1
    stream = fopen("new_debug.txt", "w");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%d %d %d %f %f %f\n", i, hm->Index[i], hm->Hash[i], hm->PosX[i], hm->PosY[i], hm->PosZ[i]);
    fclose(stream);
	*/
	/*
    // Stream file add1
    stream = fopen("new_debug.txt", "w");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%d %f %f %f %f %f %f\n", i, hm->VelX[i], hm->VelY[i], hm->VelZ[i], hm->Density[i], hm->Energy[i], hm->Pressure[i]);
    fclose(stream);
    */
    // Stream file add1
    stream = fopen("new_debug.txt", "w");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%d %d %d %d %d\n", i, hm->Index[i], hm->Hash[i], hm->SetStart[hm->Hash[i]], hm->SetStop[hm->Hash[i]]);
    fclose(stream);
    /*
    for (i = 0; i < hm->pn; i++) {
    	printf("%d - ", i);
    	for (int j = 0; j < MAXN; j++)
    		printf("%d ", hm->List[i * MAXN +j]);
    	printf("\n");
    }
    */

    return 0;
}


int outputVTK(struct model *hm, int ss) {
    /**
     * \brief Output Data file
     *
     * Saves vtk data file
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    char filename[80];
    int i;

    // Stream position file
    sprintf(filename, "out%05d.vtk", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "# vtk DataFile Version 2.0\n");
    fprintf(stream, "Unstructured Grid Example\n");
    fprintf(stream, "ASCII\n");
    fprintf(stream, "DATASET UNSTRUCTURED_GRID\n");

    fprintf(stream, "POINTS %i float\n", hm->pn);
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+e %+e %+e \n", hm->PosX[i], hm->PosY[i], hm->PosZ[i]);

    fprintf(stream, "CELLS %i %i \n", hm->pn, 2*hm->pn);
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%i %i \n", 1, i);

    fprintf(stream, "CELL_TYPES %i \n", hm->pn);
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%i \n", 1);

    fprintf(stream, "POINT_DATA %i \n", hm->pn);

    fprintf(stream, "SCALARS material int 1 \n", hm->pn);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+d \n", hm->Material[i]);

    fprintf(stream, "SCALARS density float 1 \n", hm->pn);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+e \n", hm->Density[i]);

    fprintf(stream, "SCALARS pressure float 1 \n", hm->pn);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+e \n", hm->Pressure[i]);

    fprintf(stream, "SCALARS energy float 1 \n", hm->pn);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+e \n", hm->Energy[i]);

    fprintf(stream, "VECTORS velocity float\n");
    for (i = 0; i < hm->pn; i++)
        fprintf(stream, "%+e %+e %+e \n", hm->VelX[i], hm->VelY[i], hm->VelZ[i]);

    fclose(stream);
    /*
    for (i = 0; i < hm->pn; i++) printf("%d %d \n", i, hm->Hash[i]);
    printf("\n\n\n");
    for (i = 0; i < hm->SetStart.size(); i++) printf("%d %d %d \n", i, hm->SetStart[i], hm->SetStop[i]);

    for (i = 0; i < hm->pn; i++) {
    	printf("%d  -  ", i);
    	for (j = 0; j < MAXN; j++) printf("%d ", hm->List[i*MAXN +j]);
    	printf("\n");
    }
    */
    return 0;
}


int outputCase() {
    /**
     * \brief Output Case file
     *
     * Saves ensight case file
     *
     * \date Jul 5, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    int ts;

    // Open stream file
    stream = fopen("armando.case", "w");

    fprintf(stream, "# Ensight formatted case file for Armando\n");
    fprintf(stream, "\n");
    fprintf(stream, "FORMAT\n");
    fprintf(stream, "type: ensight gold\n");
    fprintf(stream, "\n");
    fprintf(stream, "GEOMETRY\n");
    fprintf(stream, "model:    1           armando_pos_*****.geo\n");
    fprintf(stream, "\n");
    fprintf(stream, "VARIABLE\n");
    fprintf(stream, "vector per node:    1 velocity armando_vel_*****.dat\n");
    fprintf(stream, "scalar per node:    1 density  armando_rho_*****.dat\n");
    fprintf(stream, "scalar per node:    1 pressure armando_pre_*****.dat\n");
    fprintf(stream, "scalar per node:    1 energy   armando_ene_*****.dat\n");
    fprintf(stream, "\n");
    fprintf(stream, "TIME\n");
    fprintf(stream, "time set: %i\n", 1);
    fprintf(stream, "number of steps: %i\n", (hRun.tsn / hRun.ssi + 1));
    fprintf(stream, "filename start number: %i\n", 0);
    fprintf(stream, "filename increment: %i\n", 1);
    fprintf(stream, "time values:\n");

    for (ts = 0; ts <= hRun.tsn; ts++)
        if ((ts % hRun.ssi) == 0)
            fprintf(stream, "%14.8e\n", (ts * hRun.dt));

    // Close stream file
    fclose(stream);

    return 0;
}

int outputData(int ss) {
    /**
     * \brief Output Data file
     *
     * Saves ensight data file
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    char filename[80];
    int i;

    // Stream position file
    sprintf(filename, "armando_pos_%05d.geo", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "Armando output in EnSight Gold format\n");
    fprintf(stream, "EnSight 8.0.7\n");
    fprintf(stream, "node id assign\n");
    fprintf(stream, "element id assign\n");
    fprintf(stream, "extents\n");
    fprintf(stream, " 1.00000e+38-1.00000e+38\n");
    fprintf(stream, " 1.00000e+38-1.00000e+38\n");
    fprintf(stream, " 1.00000e+38-1.00000e+38\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "SPH particles\n");
    fprintf(stream, "coordinates\n");
    fprintf(stream, "%10i\n", hPN);

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hPosX[i]);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hPosY[i]);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", 0.0);

    fclose(stream);

    // Stream velocity file
    sprintf(filename, "armando_vel_%05d.dat", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "particle velocity in EnSight Gold format\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "coordinates\n");

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hVelX[i]);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hVelY[i]);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", 0.0);

    fclose(stream);

    // Stream density file
    sprintf(filename, "armando_rho_%05d.dat", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "particle density in EnSight Gold format\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "coordinates\n");

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hDensity[i]);

    fclose(stream);

    // Stream pressure file
    sprintf(filename, "armando_pre_%05d.dat", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "particle pressure in EnSight Gold format\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "coordinates\n");

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hPressure[i]);

    fclose(stream);

    // Stream energy file
    sprintf(filename, "armando_ene_%05d.dat", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "particle energy in EnSight Gold format\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "coordinates\n");

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hEnergy[i]);

    fclose(stream);

    return 0;
}


int outputVTKOld(int ss) {
    /**
     * \brief Output Data file
     *
     * Saves vtk data file
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    char filename[80];
    int i;

    // Stream position file
    sprintf(filename, "out%05d.vtk", ss);
    stream = fopen(filename, "w");
	
	fprintf(stream, "# vtk DataFile Version 2.0\n");
    fprintf(stream, "Unstructured Grid Example\n");
    fprintf(stream, "ASCII\n");
    fprintf(stream, "DATASET UNSTRUCTURED_GRID\n");

    fprintf(stream, "POINTS %i float\n", hPN);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e %+e %+e \n", hPosX[i], hPosY[i], 0.0);

    fprintf(stream, "CELLS %i %i \n", hPN, 2*hPN);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%i %i \n", 1, i);

    fprintf(stream, "CELL_TYPES %i \n", hPN);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%i \n", 1);
    
    fprintf(stream, "POINT_DATA %i \n", hPN);
    
    fprintf(stream, "SCALARS density float 1 \n", hPN);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e \n", hDensity[i]);
    
    fprintf(stream, "SCALARS pressure float 1 \n", hPN);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e \n", hPressure[i]);
    
    fprintf(stream, "SCALARS energy float 1 \n", hPN);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e \n", hEnergy[i]);
    
    fprintf(stream, "VECTORS velocity float\n");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e %+e %+e \n", hVelX[i], hVelY[i], 0.0);
    
    fclose(stream);

    return 0;
}


void initDamBreak() {

    int i, j, m, pi;
    double rho, c0, pmin;
    double dr;

    m = 1;
    rho = 1000.;
    c0 = 50.;
    pmin = -1.e12;

    hMatType[m] = 4;
    hMatProp[m][0] = rho;
    hMatProp[m][1] = c0;
    hMatProp[m][2] = pmin;

    dr = 0.02 / 4; // x4
    pi = 0;

    for (j = 0; j <= 50 * 4 ; j++) {
        for (i = 0; i <= 100 * 4; i++) {
            hPosX[pi] = i * dr + 0.5 * dr;
            hPosY[pi] = j * dr + 0.5 * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = m;
            hDensity[pi] = rho; //+ (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
            pi++;
        }
    }
    // 0 - 268   0 - 150
    /*
        for (j = 151; j <= 153; j++) {
            for (i = -3; i <= 271; i++) {
                hPosX[pi] = i * dr;
                hPosY[pi] = j * dr;

                hVelX[pi] = 0.0;
                hVelY[pi] = 0.0;
                hMaterial[pi] = -m;
                hDensity[pi] = rho; // + (9.81 * rho / c0 / c0 * (50 - j) * dr);
                hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
                pi++;
            }
        }
    */
    for (j = -3; j <= -1; j++) {
        for (i = -3; i <= 269 * 4 + 2; i++) {
            hPosX[pi] = i * dr;
            hPosY[pi] = j * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = -m;
            hDensity[pi] = rho; // + (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
            pi++;
        }
    }

    for (j = -0; j <= 80 * 4; j++) {
        for (i = -3; i <= -1; i++) {
            hPosX[pi] = i * dr;
            hPosY[pi] = j * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = -m;
            hDensity[pi] = rho; // + (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
            pi++;
        }
    }

    for (j = -0; j <= 80 * 4; j++) {
        for (i = 269 * 4; i <= 269 * 4 +2; i++) {
            hPosX[pi] = i * dr;
            hPosY[pi] = j * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = -m;
            hDensity[pi] = rho; // + (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
            pi++;
        }
    }

    hPN = pi;
    hSmooth = 1.2 * dr;
    hMass = rho * dr * dr;
    hSound = c0;

    hRun.minX = -1.0;
    hRun.maxX =  6.0;
    hRun.minY = -1.0;
    hRun.maxY =  4.0;

    hRun.dt = 4.0e-4 / 4; //1.0e-3;
    hRun.tsn = 10000 * 4; //1000;
    hRun.ssi = 200 * 4;

    hGrid.oX = hRun.minX;
    hGrid.oY = hRun.minY;
    hGrid.size = 2.0 * hSmooth;
    hGrid.nX = (int) ((hRun.maxX - hRun.minX) / hGrid.size) +1;
    hGrid.nY = (int) ((hRun.maxY - hRun.minY) / hGrid.size) +1;


    printf("Dam break in a box \n");
    printf("Particles: %i \n", hPN);
}


void initFree() {

    int i, j, m, pi;
    double rho, c0, pmin;
    double dr;

    m = 1;
    rho = 1000.;
    c0 = 50.;
    pmin = -1.e12;

    hMatType[m] = 4;
    hMatProp[m][0] = rho;
    hMatProp[m][1] = c0;
    hMatProp[m][2] = pmin;

    dr = 0.05; // x4
    pi = 0;

    for (j = 0; j < 20; j++) {
        for (i = 0; i < 20; i++) {
            hPosX[pi] = i * dr + 0.0 * dr;
            hPosY[pi] = j * dr + 0.0 * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = m;
            hDensity[pi] = rho; //+ (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 1.0;
            pi++;
        }
    }
    
    for (j = -3; j < -1; j++) {
        for (i = 0; i < 20; i++) {
            hPosX[pi] = i * dr + 0.0 * dr;
            hPosY[pi] = j * dr + 0.0 * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = -m;
            hDensity[pi] = rho; //+ (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 1.0;
            pi++;
        }
    }
    
    hPN = pi;
    hSmooth = 1.2 * dr;
    hMass = rho * dr * dr;
    hSound = c0;

    hRun.minX = -1.5;
    hRun.maxX =  2.5;
    hRun.minY = -1.5;
    hRun.maxY =  2.5;
    hRun.minZ = -1.5;
    hRun.maxZ =  2.5;

    hRun.dt = 1e-3; //1.0e-3;
    hRun.tsn = 600; //1000;
    hRun.ssi = 200;

    hGrid.oX = hRun.minX;
    hGrid.oY = hRun.minY;
    hGrid.oZ = hRun.minZ;
    hGrid.size = 2.0 * hSmooth;
    hGrid.nX = (int) ((hRun.maxX - hRun.minX) / hGrid.size) +1;
    hGrid.nY = (int) ((hRun.maxY - hRun.minY) / hGrid.size) +1;
    hGrid.nZ = (int) ((hRun.maxZ - hRun.minZ) / hGrid.size) +1;

    hLoad[0].minX = hRun.minX;
    hLoad[0].maxX = hRun.maxX;
    hLoad[0].minY = hRun.minY;
    hLoad[0].maxY = hRun.maxY;
    hLoad[0].minZ = hRun.minZ;
    hLoad[0].maxZ = hRun.maxZ;
    hLoad[0].gy = -9.81f;
    

    printf("Freefall\n");
    printf("Particles: %i \n", hPN);
}


void initBox(struct model *hm) {

    int i, j, k, m, b, pi;
    double rho, c0, pmin;
    double dr;

    m = 1;
    b = 2;
    rho = 1000.;
    c0 = 50.;
    pmin = -1.e12;

    hMatType[m] = 4;
    hMatProp[m][0] = rho;
    hMatProp[m][1] = c0;
    hMatProp[m][2] = pmin;

    hMatType[b] = 0;
    hMatProp[b][0] = rho;
    hMatProp[b][1] = c0;
    hMatProp[b][2] = pmin;

    dr = 0.1; // x4
    pi = 0;

    for (k = 0; k < 10; k++) {
		for (j = 0; j < 10; j++) {
			for (i = 0; i < 10; i++) {
				hm->PosX[pi] = i * dr + 0.0 * dr;
				hm->PosY[pi] = j * dr + 0.0 * dr;
				hm->PosZ[pi] = k * dr + 0.0 * dr;
	
				hm->VelX[pi] = 0.0;
				hm->VelY[pi] = 0.0;
				hm->VelZ[pi] = 0.0;
				hm->Material[pi] = m;
				hm->Density[pi] = rho; //+ (9.81 * rho / c0 / c0 * (50 - j) * dr);
				hm->Energy[pi] = 0.0;
				hm->Pressure[pi] = 1.0;
				pi++;
			}
        }
    }
    
    for (k = 0; k < 10; k++) {
		for (j = -2; j < -1; j++) {
			for (i = 0; i < 10; i++) {
				hm->PosX[pi] = i * dr + 0.0 * dr;
				hm->PosY[pi] = j * dr + 0.0 * dr;
				hm->PosZ[pi] = k * dr + 0.0 * dr;
				
				hm->VelX[pi] = 0.0;
				hm->VelY[pi] = 0.0;
				hm->VelZ[pi] = 0.0;
				hm->Material[pi] = b;
				hm->Density[pi] = rho; //+ (9.81 * rho / c0 / c0 * (50 - j) * dr);
				hm->Energy[pi] = 0.0;
				hm->Pressure[pi] = 1.0;
				pi++;
			}
        }
    }
    
    hPN = pi;
    hSmooth = 1.2 * dr;
    hMass = rho * dr * dr * dr;
    hSound = c0;

    hRun.minX = -2.5;
    hRun.maxX =  2.5;
    hRun.minY = -2.5;
    hRun.maxY =  2.5;
    hRun.minZ = -2.5;
    hRun.maxZ =  2.5;

    hRun.dt = 2.0e-3; //1.0e-3;
    hRun.tsn = 600; //1000;
    hRun.ssi = 200;

    hGrid.oX = hRun.minX;
    hGrid.oY = hRun.minY;
    hGrid.oZ = hRun.minZ;
    hGrid.size = 2.0 * hSmooth;
    hGrid.nX = (int) ((hRun.maxX - hRun.minX) / hGrid.size) +1;
    hGrid.nY = (int) ((hRun.maxY - hRun.minY) / hGrid.size) +1;
    hGrid.nZ = (int) ((hRun.maxZ - hRun.minZ) / hGrid.size) +1;

    hLoad[0].minX = hRun.minX;
    hLoad[0].maxX = hRun.maxX;
    hLoad[0].minY = hRun.minY;
    hLoad[0].maxY = hRun.maxY;
    hLoad[0].minZ = hRun.minZ;
    hLoad[0].maxZ = hRun.maxZ;
    hLoad[0].gy = -9.81f;
    
	hm->pn = hPN;
	for (i = 0; i < hPN; i++) {
		hm->Mass[i] = hMass;
		hm->Smooth[i] = hSmooth;
	}

    printf("Freefall\n");
    printf("Particles: %i \n", hPN);
}



int iSort(int *array, int *perm, int n) {
    int i;
    static int* dummy = NULL;

    if (!dummy) dummy = (int *) malloc(MAXP * sizeof(int));

    for (i = 0; i < n; i++) dummy[i] = array[i];
    for (i = 0; i < n; i++) array[i] = dummy[perm[i]];

    return 0;
}

int fSort(float *array, int *perm, int n) {
    int i;
    static float* dummy = NULL;

    if (!dummy) dummy = (float *) malloc(MAXP * sizeof(float));

    for (i = 0; i < n; i++) dummy[i] = array[i];
    for (i = 0; i < n; i++) array[i] = dummy[perm[i]];

    return 0;
}


int sortArraysHost(void) {

    // Particles are re ordered
    
    //iSort(hHash, hIndex, hPN);
    iSort(hMaterial, hIndex, hPN);
    fSort(hPosX, hIndex, hPN);
    fSort(hPosY, hIndex, hPN);
    fSort(hVelX, hIndex, hPN);
    fSort(hVelY, hIndex, hPN);
    fSort(hDensity, hIndex, hPN);
    fSort(hEnergy, hIndex, hPN);
    fSort(hPressure, hIndex, hPN);
    
    return 0;
}


int indexCompare(const void *a, const void *b)
{
	int c, i1, i2;
	c = 0;
	i1 = *(int*)a;
	i2 = *(int*)b;
	if (hHash[i1] < hHash[i2]) c = -1;
	if (hHash[i1] > hHash[i2]) c = 1;
  return c;
}

int mapCompare(const void *a, const void *b)
{
	int c;
	struct pair m1, m2;
	
	c = 0;
	m1 = *(struct pair*)a;
	m2 = *(struct pair*)b;
	if (m1.key < m2.key) c = -1;
	if (m1.key > m2.key) c = 1;
  return c;
}



int updateHashHostOld() {
	
    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

	int ip, ix, iy, ic;
	
    for (ip = 0; ip < hPN; ip++) {
        ix = (int) ((hPosX[ip] - hGrid.oX) / hGrid.size);
        iy = (int) ((hPosY[ip] - hGrid.oY) / hGrid.size);
		ic = ix + iy * hGrid.nX;
		
		hHash[ip] = ic;
		hIndex[ip] = ip;
	}
	
	return 0;
}

void updateHashHost(const int pn, const struct grid Grid,
                             const float* PosX, const float* PosY, const float* PosZ,
                             int* Hash) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    int ip, ix, iy, iz, ic;

    for (ip = 0; ip < pn; ip++) {
        ix = (int) truncf((PosX[ip] - Grid.oX) / Grid.size);
        iy = (int) truncf((PosY[ip] - Grid.oY) / Grid.size);
        iz = (int) truncf((PosZ[ip] - Grid.oZ) / Grid.size);
        ic = ix + iy * Grid.nX + iz * Grid.nX * Grid.nY;
		
		if (ic < 0) ic = 0;
		if (ic > Grid.nX * Grid.nY * Grid.nZ) ic = ic = Grid.nX * Grid.nY * Grid.nZ;
        Hash[ip] = ic;
    }
}


__global__ void updateHashDevice(const int pn, const struct grid Grid,
                                 const float* PosX, const float* PosY, const float* PosZ,
                                 int* Hash) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    int ip, ix, iy, iz, ic;

    ip = threadIdx.x + blockDim.x * blockIdx.x;

    if (ip < pn) {
        ix = (int) truncf((PosX[ip] - Grid.oX) / Grid.size);
        iy = (int) truncf((PosY[ip] - Grid.oY) / Grid.size);
        iz = (int) truncf((PosZ[ip] - Grid.oZ) / Grid.size);
        ic = ix + iy * Grid.nX + iz * Grid.nX * Grid.nY;

		if (ic < 0) ic = 0;
		if (ic > Grid.nX * Grid.nY * Grid.nZ) ic = ic = Grid.nX * Grid.nY * Grid.nZ;
        Hash[ip] = ic;
    }
}

int updateSetsHostOld() {
	
    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */
	
	int ip;
	
	hSetStart[hHash[0]] = 0;
	if (hHash[0] != hHash[1]) hSetStop[hHash[0]] = 1;

    for (ip = 1; ip < hPN -1; ip++) {
		if (hHash[ip] != hHash[ip -1]) hSetStart[hHash[ip]] = ip;
		if (hHash[ip] != hHash[ip +1]) hSetStop[hHash[ip]] = ip +1;
	}
	
	if (hHash[hPN -1] != hHash[hPN -2]) hSetStart[hHash[hPN -1]] = hPN -1;
	hSetStop[hHash[hPN -1]] = hPN;
	
	return 0;
}


void updateSetsHost(const int pn, int *SetStart, int *SetStop,
                               const int* Hash) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    int ip;
    int hash, nextHash, prevHash;
    
    for (ip = 0; ip < pn; ip++) {
		hash = Hash[ip];
		if (ip == 0) prevHash = -1;
		else prevHash = Hash[ip -1];
		if (ip == pn -1) nextHash = -1;
		else nextHash = Hash[ip +1];
		
		if (hash != prevHash) SetStart[hash] = ip;
		if (hash != nextHash) SetStop[hash] = ip +1;
	}

}


__global__ void updateSetsDevice(const int pn, int *SetStart, int *SetStop,
                                 const int* Hash) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    __shared__ int prevHash[THREADS];
    __shared__ int nextHash[THREADS];

    int ip;
    int hash;

    ip = threadIdx.x + blockDim.x * blockIdx.x;
    if (ip >= pn) return;

    hash = Hash[ip];

    if (threadIdx.x < THREADS -1) prevHash[threadIdx.x +1] = hash;
    if (threadIdx.x > 0) nextHash[threadIdx.x -1] = hash;

    if (threadIdx.x == 0) {
        if (ip == 0) prevHash[threadIdx.x] = -1;
        else prevHash[threadIdx.x] = Hash[ip -1];
    }

    if (threadIdx.x == THREADS -1) {
        if (ip == pn -1) nextHash[threadIdx.x] = -1;
        else nextHash[threadIdx.x] = Hash[ip +1];
    }

    __syncthreads();

    if (hash != prevHash[threadIdx.x]) SetStart[hash] = ip;

    if (hash != nextHash[threadIdx.x]) SetStop[hash] = ip +1;

}


int updateListHostOld(void) {
	int ip, ic, ix, iy, il, i, j, jp, jc, np;
	float dx, dy, dr;
	
    // Particles list is filled
    for (ip = 0; ip < hPN; ip++) {
		for (il = 0; il < MAXN; il++) {
			hList[ip * MAXN + il] = ip;
		}
		
        ix = (int) ((hPosX[ip] - hGrid.oX) / hGrid.size);
        iy = (int) ((hPosY[ip] - hGrid.oY) / hGrid.size);
		ic = ix + iy * hGrid.nX;
		
		np = 0;
        for (j = -1; j <= 1; j++) {
            for (i = -1; i <= 1; i++) {
				jc = ic + i + j * hGrid.nX;
				
				for (jp = hSetStart[jc]; jp < hSetStop[jc]; jp++) {
					dx = hPosX[ip] - hPosX[jp];
                    dy = hPosY[ip] - hPosY[jp];
                    dr = sqrtf(dx * dx + dy * dy);
					
					if ((dr < 2.0 * hSmooth) && (np < MAXN)) {
						hList[ip * MAXN + np] = jp;
						np++;
					}
				}
			}
		}
		
		
	}
	
	return 0;
}


void updateListHost(const int pn, int *List,
                             const int* SetStart, const int* SetStop,
                             const struct grid Grid, const float* Smooth,
                             const float* PosX, const float* PosY, const float* PosZ) {

    int ip, ic, ix, iy, iz, i, j, k, jp, jc, np;
    float dx, dy, dz, dr;

    // Particles list is filled
    for (ip = 0; ip < pn; ip++) {
		ix = (int) ((PosX[ip] - Grid.oX) / Grid.size);
		iy = (int) ((PosY[ip] - Grid.oY) / Grid.size);
		iz = (int) ((PosZ[ip] - Grid.oZ) / Grid.size);
		ic = ix + iy * Grid.nX + iz * Grid.nX * Grid.nY;
		np = 0;
		
		for (k = -1; k <= 1; k++) {
			for (j = -1; j <= 1; j++) {
				for (i = -1; i <= 1; i++) {
					jc = ic + i + j * Grid.nX + k * Grid.nX * Grid.nY;
					
					if (jc >= 0 && jc <= Grid.nX * Grid.nY * Grid.nZ) {
						for (jp = SetStart[jc]; jp < SetStop[jc]; jp++) {
							dx = PosX[ip] - PosX[jp];
							dy = PosY[ip] - PosY[jp];
							dz = PosZ[ip] - PosZ[jp];
							dr = sqrtf(dx * dx + dy * dy + dz * dz);
							
							if ((dr < 2.0f * Smooth[ip]) && (np < MAXN)) {
								List[ip * MAXN + np] = jp;
								np++;
							}
						}
					}
				}
			}
		}
		
		while (np < MAXN) {
			List[ip * MAXN + np] = ip;
			np++;
		}
	}
}


__global__ void updateListDevice(const int pn, int *List,
                                 const int* SetStart, const int* SetStop,
                                 const struct grid Grid, const float* Smooth,
                                 const float* PosX, const float* PosY, const float* PosZ) {

    int ip, ic, ix, iy, iz, i, j, k, jp, jc, np;
    float dx, dy, dz, dr;

    // Particles list is filled
    ip = threadIdx.x + blockDim.x * blockIdx.x;
    if (ip >= pn) return;

    ix = (int) ((PosX[ip] - Grid.oX) / Grid.size);
    iy = (int) ((PosY[ip] - Grid.oY) / Grid.size);
    iz = (int) ((PosZ[ip] - Grid.oZ) / Grid.size);
    ic = ix + iy * Grid.nX + iz * Grid.nX * Grid.nY;
    np = 0;

    for (k = -1; k <= 1; k++) {
        for (j = -1; j <= 1; j++) {
            for (i = -1; i <= 1; i++) {
                jc = ic + i + j * Grid.nX + k * Grid.nX * Grid.nY;
				
				if (jc >= 0 && jc <= Grid.nX * Grid.nY * Grid.nZ) {
					for (jp = SetStart[jc]; jp < SetStop[jc]; jp++) {
						dx = PosX[ip] - PosX[jp];
						dy = PosY[ip] - PosY[jp];
						dz = PosZ[ip] - PosZ[jp];
						dr = sqrtf(dx * dx + dy * dy + dz * dz);
						
						if ((dr < 2.0f * Smooth[ip]) && (np < MAXN)) {
							List[ip * MAXN + np] = jp;
							np++;
						}
					}
                }
            }
        }
    }

    while (np < MAXN) {
        List[ip * MAXN + np] = ip;
        np++;
    }

}

int neighbourListHost(struct model *hm) {
	struct pair map[MAXP];
	int i, ip;
	
	//updateHashHostOld();
    updateHashHost(hm->pn, hGrid, hm->PosX, hm->PosY, hm->PosZ, hm->Hash);
    for (ip = 0; ip < hm->pn; ip++) hm->Index[ip] = ip;
	
	//qsort(hIndex, hPN, sizeof(int), indexCompare);
    for (ip = 0; ip < hm->pn; ip++) {
		map[ip].key = hm->Hash[ip];
		map[ip].value = hm->Index[ip];
	}
	qsort(map, hm->pn, sizeof(struct pair), mapCompare);
    for (ip = 0; ip < hm->pn; ip++) {
		hm->Hash[ip] = map[ip].key;
		hm->Index[ip] = map[ip].value;
	}
	
	//sortArraysHost();
    iSort(hMaterial, hIndex, hPN);
    fSort(hPosX, hIndex, hPN);
    fSort(hPosY, hIndex, hPN);
    fSort(hVelX, hIndex, hPN);
    fSort(hVelY, hIndex, hPN);
    fSort(hDensity, hIndex, hPN);
    fSort(hEnergy, hIndex, hPN);
    fSort(hPressure, hIndex, hPN);
    
    fSort(hm->Mass, hm->Index, hm->pn);
    fSort(hm->Smooth, hm->Index, hm->pn);
    fSort(hm->PosZ, hm->Index, hm->pn);
    fSort(hm->VelZ, hm->Index, hm->pn);
	
	for (i = 0; i < hGrid.nX * hGrid.nY * hGrid.nZ; i++) hm->SetStart[i] = 0;
	for (i = 0; i < hGrid.nX * hGrid.nY * hGrid.nZ; i++) hm->SetStop[i] = 0;
	
	updateSetsHost(hm->pn, hm->SetStart, hm->SetStop, hm->Hash);
	
	//updateSetsHostOld();
	
	updateListHost(hm->pn, hm->List, hm->SetStart, hm->SetStop, hGrid, hm->Smooth,
				   hm->PosX, hm->PosY, hm->PosZ);
	//updateListHostOld();
	
	return 0;
}

int neighbourListDevice(struct model *dm) {
    int blocks, threads;

    blocks = (dm->pn + THREADS - 1) / THREADS;
    threads = THREADS;
    
	thrust::device_ptr<int> tIndex(dm->Index);
	thrust::device_ptr<int> tHash(dm->Hash);
	thrust::device_ptr<int> tMaterial(dm->Material);
	thrust::device_ptr<float> tMass(dm->Mass);
	thrust::device_ptr<float> tSmooth(dm->Smooth);
	thrust::device_ptr<float> tPosX(dm->PosX);
	thrust::device_ptr<float> tPosY(dm->PosY);
	thrust::device_ptr<float> tPosZ(dm->PosZ);
	thrust::device_ptr<float> tVelX(dm->VelX);
	thrust::device_ptr<float> tVelY(dm->VelY);
	thrust::device_ptr<float> tVelZ(dm->VelZ);
	thrust::device_ptr<float> tDensity(dm->Density);
	thrust::device_ptr<float> tEnergy(dm->Energy);
	thrust::device_ptr<int> tIntDummy(dm->IntDummy);
	thrust::device_ptr<float> tFloatDummy(dm->FloatDummy);
	thrust::device_ptr<int> tSetStart(dm->SetStart);
	thrust::device_ptr<int> tSetStop(dm->SetStop);
	
	updateHashDevice <<< blocks, threads >>>
	(dm->pn, dGrid, dm->PosX, dm->PosY, dm->PosZ, dm->Hash);
	
	thrust::sequence(tIndex, tIndex + dm->pn, 1);
	
	thrust::sort_by_key(tHash, tHash + dm->pn, tIndex);
	
	thrust::copy(tMaterial, tMaterial + dm->pn, tIntDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tIntDummy, tMaterial);
	thrust::copy(tMass, tMass + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tMass);
	thrust::copy(tSmooth, tSmooth + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tSmooth);
	thrust::copy(tPosX, tPosX + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tPosX);
	thrust::copy(tPosY, tPosY + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tPosY);
	thrust::copy(tPosZ, tPosZ + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tPosZ);
	thrust::copy(tVelX, tVelX + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tVelX);
	thrust::copy(tVelY, tVelY + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tVelY);
	thrust::copy(tVelZ, tVelZ + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tVelZ);
	thrust::copy(tDensity, tDensity + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tDensity);
	thrust::copy(tEnergy, tEnergy + dm->pn, tFloatDummy);
	thrust::gather(tIndex, tIndex + dm->pn, tFloatDummy, tEnergy);
	
	thrust::fill(tSetStart, tSetStart + dm->pn, 0);
	thrust::fill(tSetStop, tSetStop + dm->pn, 0);
	
	updateSetsDevice <<< blocks, threads >>>
	(dm->pn, dm->SetStart, dm->SetStop, dm->Hash);
	
	updateListDevice <<< blocks, threads >>>
	(dm->pn, dm->List, dm->SetStart, dm->SetStop, dGrid, dm->Smooth,
	dm->PosX, dm->PosY, dm->PosZ);
	
    return 0;
}

int RKstepHost(struct model *hm, float alpha) {
	int ip;
	
	for (ip = 0; ip < hm->pn; ip++) {
		hm->VelDotX[ip] = 0.0f;
		hm->VelDotY[ip] = 0.0f;
		hm->VelDotZ[ip] = 0.0f;
		hm->DensityDot[ip] = 0.0f;
		hm->EnergyDot[ip] = 0.0f;
	}
	
    // External loads
    updateLoadsHost(hm->pn, hm->Material, 
                    hm->PosX, hm->PosY, hm->PosZ,
                    hm->VelX, hm->VelY, hm->VelZ, 
                    hm->VelDotX, hm->VelDotY, hm->VelDotZ, hm->EnergyDot);
	
	// External forces
	//updateForcesHost();
	
	// Calculate particle interactions
    balanceMassMomentumHost(hm->pn, hm->List, hm->Material, hm->Mass, hm->Smooth, 
                            hm->PosX, hm->PosY, hm->PosZ, 
                            hm->VelX, hm->VelY, hm->VelZ, 
                            hm->Density, hm->Pressure, hm->Sound, 
                            hm->DensityDot, hm->VelDotX, hm->VelDotY, hm->VelDotZ);
	//balanceMassMomentumHostOld();
	
    balanceEnergyHost(hm->pn, hm->Pressure, hm->Density, 
                      hm->DensityDot, hm->EnergyDot);
	//balanceEnergyHostOld();
	
    // Update particles
    updateParticlesHost(hm->pn, alpha, hm->Material, 
                        hm->VelDotX, hm->VelDotY, hm->VelDotZ, hm->DensityDot, hm->EnergyDot,
                        hm->PosX0, hm->PosY0, hm->PosZ0, 
                        hm->VelX0, hm->VelY0, hm->VelZ0, hm->Density0, hm->Energy0, 
                        hm->PosX, hm->PosY, hm->PosZ, hm->VelX, hm->VelY, hm->VelZ, 
                        hm->Density, hm->Energy, hm->Pressure, hm->Sound);
	//updateParticlesHostOld(alpha);
	
	return 0;
}

int RKstepDevice(struct model *dm, float alpha) {
    int blocks, threads;

    blocks = (dm->pn + THREADS - 1) / THREADS;
    threads = THREADS;
	
	thrust::device_ptr<float> tVelDotX(dm->VelDotX);
	thrust::device_ptr<float> tVelDotY(dm->VelDotY);
	thrust::device_ptr<float> tVelDotZ(dm->VelDotZ);
	thrust::device_ptr<float> tDensityDot(dm->DensityDot);
	thrust::device_ptr<float> tEnergyDot(dm->EnergyDot);
	
	thrust::fill(tVelDotX, tVelDotX + dm->pn, 0.0f);
	thrust::fill(tVelDotY, tVelDotY + dm->pn, 0.0f);
	thrust::fill(tVelDotZ, tVelDotZ + dm->pn, 0.0f);
	thrust::fill(tDensityDot, tDensityDot + dm->pn, 0.0f);
	thrust::fill(tEnergyDot, tEnergyDot + dm->pn, 0.0f);
	
	// External loads
	updateLoadsDevice <<< blocks, threads >>>
	(dm->pn, dm->Material, dm->PosX, dm->PosY, dm->PosZ, 
	dm->VelDotX, dm->VelDotY, dm->VelDotZ, dm->EnergyDot);
	
	// Calculate particle interactions
	balanceMassMomentumDevice <<< blocks, threads >>>
	(dm->pn, dm->List, dm->Material, dm->Mass, dm->Smooth, dm->PosX, dm->PosY, dm->PosZ,
	dm->VelX, dm->VelY, dm->VelZ, dm->Density, dm->Pressure, dm->Sound,
	dm->DensityDot, dm->VelDotX, dm->VelDotY, dm->VelDotZ);
	
	balanceEnergyDevice <<< blocks, threads >>>
	(dm->pn, dm->Pressure, dm->Density, dm->DensityDot, dm->EnergyDot);
	
	// Update particles
	updateParticlesDevice  <<< blocks, threads >>>
	(dm->pn, alpha, dm->Material, dm->VelDotX, dm->VelDotY, dm->VelDotZ, dm->DensityDot, dm->EnergyDot,
	dm->PosX0, dm->PosY0, dm->PosZ0, dm->VelX0, dm->VelY0, dm->VelZ0, dm->Density0, dm->Energy0,
	dm->PosX, dm->PosY, dm->PosZ, dm->VelX, dm->VelY, dm->VelZ, dm->Density, dm->Energy, dm->Pressure, dm->Sound);
	
    return 0;
}


int RKintegrateHost(struct model *hm) {

    /**
     * \brief Runge Kutta 3rd order time integration
     *
     * Integrate the Navier Stokes equations in time with the
     * Total Variation Diminishing Runge-Kutta algorithm of the 3rd order
     *
     * \date Dec 20, 2010
     * \author Luca Massidda
     */

    int ts;
	
    // TIME CYCLE
    for (ts = 0; ts <= hRun.tsn; ts++) {
		
        // Output data
        if ((ts % hRun.ssi) == 0) {
            printf("Saving time: %g \n", ts * hRun.dt);
            printData(hm);
            outputVTK(hm, ts / hRun.ssi);
        }
        
		// Calculate neighbour list
		neighbourListHost(hm);
		
		// Save initial condition
		backupDataHost(hm);
		
        // Step 1
		RKstepHost(hm, 1.0);
		
        // Step 2
		RKstepHost(hm, 1.0 / 4.0);
		
        // Step 3
		RKstepHost(hm, 2.0 / 3.0);
		
	}
	
	return 0;
}


int RKintegrateDevice(struct model *hm, struct model *dm) {

    /**
     * \brief Runge Kutta 3rd order time integration
     *
     * Integrate the Navier Stokes equations in time with the
     * Total Variation Diminishing Runge-Kutta algorithm of the 3rd order
     *
     * \date Dec 20, 2010
     * \author Luca Massidda
     */

    int ts;
	
    // TIME CYCLE
    for (ts = 0; ts <= hRun.tsn; ts++) {
		
        // Output data
        if ((ts % hRun.ssi) == 0) {
            copyDeviceToHost(dm, hm);
            printf("Saving time: %g \n", ts * hRun.dt);
            printData(hm);
            outputVTK(hm, ts / hRun.ssi);
        }
        
		// Calculate neighbour list
		neighbourListDevice(dm);
		
		// Save initial condition
		backupDataDevice(dm);
		
        // Step 1
		RKstepDevice(dm, 1.0);
		
        // Step 2
		RKstepDevice(dm, 1.0 / 4.0);
		
        // Step 3
		RKstepDevice(dm, 2.0 / 3.0);
		
	}
	
	return 0;
}


int main() {
    /**
     * \brief armando2D v2.0
     *
     * An SPH code for non stationary fluid dynamics.
     * This is the reviewed and improved C version of Armando v1.0
     * developed at CERN in 2008
     *
     * \date Oct 20, 2010
     * \author Luca Massidda
     */

    struct model hModel, dModel;
    int i;
    
    initHost(&hModel);
    for (i = 0; i < 10; i++) {
        hLoad[i].gx = 0.0f;
        hLoad[i].gy = 0.0f;
        hLoad[i].gz = 0.0f;
        hLoad[i].w = 0.0f;

        hOut[i].nX = 0.0f;
        hOut[i].nY = 0.0f;
        hOut[i].nZ = 0.0f;
    }

    initBox(&hModel);
	
    initDevice(&dModel);
    copyHostToDevice(&hModel, &dModel);
	//RKintegrateDevice(&hModel, &dModel);
	
	RKintegrateHost(&hModel);
	
    return 0;
}
