#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <cutil_inline.h>
#include <cudpp.h>

#define MAXP 100000
#define MAXN 21
#define MAXG 1280000

#define THREADS 256
#define ParticlesInSet 12
#define SetsInBlock 20

struct  grid {
    float oX, oY;
    float size;
    int nX, nY;
    int SN;
    int* set;
    int* nump;
    int* cell;
    int* perm;
};

struct  simulation {
    float minX, maxX;
    float minY, maxY;
    float dt;
    int tsn;
    int ssi;
    int nsi;
};


// Host Variables

int *hMaterial;
float *hPosX;
float *hPosY;
float *hVelX;
float *hVelY;
float *hDensity;
float *hEnergy;
float *hPressure;
float *hVelDotX;
float *hVelDotY;
float *hDensityDot;
float *hEnergyDot;
int *hList;
int *hHash;
int *hIndex;
int *hSetStart;
int *hSetStop;

int hPN;
float hSmooth, hMass, hSound;
int hMatType[10];
float hMatProp[10][10];
struct simulation hRun;
struct grid hGrid;

float *hPosX0;
float *hPosY0;
float *hVelX0;
float *hVelY0;
float *hDensity0;
float *hEnergy0;

CUDPPHandle hSortHandle;


// Device Variables

__device__ int *dMaterial;
__device__ float *dPosX;
__device__ float *dPosY;
__device__ float *dVelX;
__device__ float *dVelY;
__device__ float *dDensity;
__device__ float *dEnergy;
__device__ float *dPressure;
__device__ float *dVelDotX;
__device__ float *dVelDotY;
__device__ float *dDensityDot;
__device__ float *dEnergyDot;
__device__ int *dList;
__device__ int *dHash;
__device__ int *dIndex;
__device__ int *dSetStart;
__device__ int *dSetStop;

__device__ int *dIntDummy;
__device__ float *dFloatDummy;

__device__ __constant__ int dPN;
__device__ __constant__ float dSmooth, dMass, dSound;
__device__ __constant__ int dMatType[10];
__device__ __constant__ float dMatProp[10][10];
__device__ __constant__ struct simulation dRun;
__device__ struct grid dGrid;

__device__ float *dPosX0;
__device__ float *dPosY0;
__device__ float *dVelX0;
__device__ float *dVelY0;
__device__ float *dDensity0;
__device__ float *dEnergy0;


// Device code


__device__ float kernelWendland(float r, float h) {

    float q, alpha, w;
    /**
     * \brief Wendland kernel
     *
     * \date Feb 8, 2011
     * \author Luca Massidda
     */
	
	q = r / h;

    // for 3D
	//alpha = 21.0 / (16.0 * M_PI * h * h * h);
	
    // for 2D
	alpha = 7.0 / (4.0 * M_PI * h * h);
	
    w = 0.0;
    if (q < 2) {
        w = 1.0 - 0.5*q;
        w *= w;
        w *= w;
        w *= 1.0 + 2.0*q;
        w *= alpha;
    }

    return w;
}


__device__ float kernelDerivWendland(float r, float h) {

    float q, alpha, dwdr;
    /**
     * \brief Wendland kernel derivative
     *
     * \date Feb 8, 2011
     * \author Luca Massidda
     */

	q = r / h;
	
    // for 3D
	//alpha = 21.0 / (16.0 * M_PI * h * h * h);
	
    // for 2D
	alpha = 7.0 / (4.0 * M_PI * h * h);
	
    dwdr = 0;
    if (q < 2) {
        dwdr = 5.0 / 8.0 * q * pow((q - 2.0), 3) ;
        dwdr *= alpha / h;
    }

    return dwdr;
}


float kernelDerivWendlandHost(float r, float h) {

    float q, alpha, dwdr;
    /**
     * \brief Wendland kernel derivative
     *
     * \date Feb 8, 2011
     * \author Luca Massidda
     */

	q = r / h;
	
    // for 3D
	//alpha = 21.0 / (16.0 * M_PI * h * h * h);
	
    // for 2D
	alpha = 7.0 / (4.0 * M_PI * h * h);
	
    dwdr = 0;
    if (q < 2) {
        dwdr = 5.0 / 8.0 * q * pow((q - 2.0), 3) ;
        dwdr *= alpha / h;
    }

    return dwdr;
}


__device__ float kernelGauss(float r, float h) {

    float r2, q2, h2, alpha, w;//, dwdr;
    /**
     * \brief Gauss kernel
     *
     * \date Dec 21, 2010
     * \author Luca Massidda
     */

    r2 = r * r ;
    h2 = h * h;
    q2 = r2 / h2;


    //alpha = 1.0 / (pow(h, 1) * pow(3.14, 0.5));
    alpha = 1.0 / (3.14 * h2);

    w = 0.0;
    //dwdr = 0.0;

    if (q2 < 4.0) {
        w = alpha * expf(-q2);
        //dwdr = w * (-2.0 * r / h2);
    }

    return w;
}


__device__ float kernelDerivGauss(float r, float h) {

    float r2, q2, h2, alpha, w, dwdr;
    /**
     * \brief Gauss kernel
     *
     * \date Dec 21, 2010
     * \author Luca Massidda
     */

    r2 = r * r ;
    h2 = h * h;
    q2 = r2 / h2;


    //alpha = 1.0 / (pow(h, 1) * pow(3.14, 0.5));
    alpha = 1.0 / (3.14 * h2);

    w = 0.0;
    dwdr = 0.0;

    if (q2 < 4.0) {
        w = alpha * expf(-q2);
        dwdr = w * (-2.0 * r / h2);
    }

    return dwdr;
}


__device__ float pressureGas(int mat ,float rho, float u) {
    /**
     * \brief Ideal gas Equation Of State
     *
     * p = (k -1) rho u
     * c = (k(k -1) u)^0.5
     *
     * k = dMatProp[mat][1]
     * pshift = dMatProp[mat][2]
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float p;
//    float c;

    p = (dMatProp[mat][1] - 1.0) * rho * u;
    p += dMatProp[mat][2];

//    c = sqrtf(dMatProp[mat][1] * (dMatProp[mat][1] - 1.0) * u);

    return p;
}



__device__ float pressurePoly(int mat , float rho, float u) {
    /**
     * \brief Mie-Gruneisen polynomial Equation Of State
     *
     * p = a1 mu + a2 mu^2 + a3 mu^3 + (b0 + b1 mu) rho0 u  in compression
     * p = t1 mu + t2 mu^2 + b0 rho0 u                      in tension
     *
     * rho0 = dMatProp[mat][0];
     * a1 = dMatProp[mat][1];
     * a2 = dMatProp[mat][2];
     * a3 = dMatProp[mat][3];
     * b0 = dMatProp[mat][4];
     * b1 = dMatProp[mat][5];
     * t1 = dMatProp[mat][6];
     * t2 = dMatProp[mat][7];
     * pmin = dMatProp[mat][8];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float mu;
    float p;
//    float c;

    mu = (rho - dMatProp[mat][0]) / dMatProp[mat][0];

    if (mu < 0)
        p = (dMatProp[mat][6] * mu + dMatProp[mat][7] * mu*mu)
            + (dMatProp[mat][4] * dMatProp[mat][0] * u);
    else
        p = (dMatProp[mat][1] * mu + dMatProp[mat][2] * mu*mu
             + dMatProp[mat][3] * mu*mu*mu)
            + ((dMatProp[mat][4] + dMatProp[mat][5] * mu)
               * dMatProp[mat][0] * u);

    if (p < dMatProp[mat][8]) p = dMatProp[mat][8];

//    c = sqrtf(dMatProp[mat][1] / rho);

    return p;
}

__device__ float pressureShock(int mat, float rho, float u) {
    /**
     * \brief Mie-Gruneisen Shock Hugoniot Equation Of State
     *
     * mu = rho / rho0 -1
     * g = g * rho0 / rho
     * ph = (rho0 c0^2 mu (1 + mu)) / (1 - (s0 - 1) * mu)^2
     * uh = 1/2 ph/rho0 * (mu / (1 + mu))
     * p = ph + g * rho * (u - uh)
     *
     * rho0 = dMatProp[mat][0];
     * c0 = dMatProp[mat][1];
     * g0 = dMatProp[mat][2];
     * s0 = dMatProp[mat][3];
     * pmin = dMatProp[mat][4];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float mu;
    float p, ph;
//    float c;

    mu = (rho - dMatProp[mat][0]) / dMatProp[mat][0];

    ph = (dMatProp[mat][0] * powf(dMatProp[mat][1], 2) * mu*(1.0 +mu))
         / powf((1.0 - (dMatProp[mat][3] -1.0) * mu), 2);

    p = ph + dMatProp[mat][2] * dMatProp[mat][0]
        * (u - (0.5 * ph / dMatProp[mat][0] * (mu / (1.0 + mu))));

    if (p < dMatProp[mat][4]) p = dMatProp[mat][4];

//    c = dMatProp[mat][1];

    return p;
}


__device__ float pressureTait(int mat, float rho, float u) {
    /**
     * \brief Tait Equation Of State
     *
     * p = rho0 * c0 * c0 / 7.0 * (powf((rho / rho0), 7) - 1.0);
     * c = c0;
     *
     * rho0 = dMatProp[mat][0];
     * c0 = dMatProp[mat][1];
     * pmin = dMatProp[mat][2];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float p;
//    float c;

    p = dMatProp[mat][0] * powf(dMatProp[mat][1], 2) / 7.0
        * (powf((rho / dMatProp[mat][0]), 7) - 1.0);

    if (p < dMatProp[mat][2]) p = dMatProp[mat][2];

//    c = dMatProp[mat][1];

    return p;
}


float pressureGasHost(int mat ,float rho, float u) {
    /**
     * \brief Ideal gas Equation Of State
     *
     * p = (k -1) rho u
     * c = (k(k -1) u)^0.5
     *
     * k = dMatProp[mat][1]
     * pshift = dMatProp[mat][2]
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float p;
//    float c;

    p = (hMatProp[mat][1] - 1.0) * rho * u;
    p += hMatProp[mat][2];

//    c = sqrtf(hMatProp[mat][1] * (hMatProp[mat][1] - 1.0) * u);

    return p;
}



float pressurePolyHost(int mat , float rho, float u) {
    /**
     * \brief Mie-Gruneisen polynomial Equation Of State
     *
     * p = a1 mu + a2 mu^2 + a3 mu^3 + (b0 + b1 mu) rho0 u  in compression
     * p = t1 mu + t2 mu^2 + b0 rho0 u                      in tension
     *
     * rho0 = dMatProp[mat][0];
     * a1 = dMatProp[mat][1];
     * a2 = dMatProp[mat][2];
     * a3 = dMatProp[mat][3];
     * b0 = dMatProp[mat][4];
     * b1 = dMatProp[mat][5];
     * t1 = dMatProp[mat][6];
     * t2 = dMatProp[mat][7];
     * pmin = dMatProp[mat][8];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float mu;
    float p;
//    float c;

    mu = (rho - hMatProp[mat][0]) / hMatProp[mat][0];

    if (mu < 0)
        p = (hMatProp[mat][6] * mu + hMatProp[mat][7] * mu*mu)
            + (hMatProp[mat][4] * hMatProp[mat][0] * u);
    else
        p = (hMatProp[mat][1] * mu + hMatProp[mat][2] * mu*mu
             + hMatProp[mat][3] * mu*mu*mu)
            + ((hMatProp[mat][4] + hMatProp[mat][5] * mu)
               * hMatProp[mat][0] * u);

    if (p < hMatProp[mat][8]) p = hMatProp[mat][8];

//    c = sqrtf(hMatProp[mat][1] / rho);

    return p;
}

float pressureShockHost(int mat, float rho, float u) {
    /**
     * \brief Mie-Gruneisen Shock Hugoniot Equation Of State
     *
     * mu = rho / rho0 -1
     * g = g * rho0 / rho
     * ph = (rho0 c0^2 mu (1 + mu)) / (1 - (s0 - 1) * mu)^2
     * uh = 1/2 ph/rho0 * (mu / (1 + mu))
     * p = ph + g * rho * (u - uh)
     *
     * rho0 = dMatProp[mat][0];
     * c0 = dMatProp[mat][1];
     * g0 = dMatProp[mat][2];
     * s0 = dMatProp[mat][3];
     * pmin = dMatProp[mat][4];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float mu;
    float p, ph;
//    float c;

    mu = (rho - hMatProp[mat][0]) / hMatProp[mat][0];

    ph = (hMatProp[mat][0] * powf(hMatProp[mat][1], 2) * mu*(1.0 +mu))
         / powf((1.0 - (hMatProp[mat][3] -1.0) * mu), 2);

    p = ph + hMatProp[mat][2] * hMatProp[mat][0]
        * (u - (0.5 * ph / hMatProp[mat][0] * (mu / (1.0 + mu))));

    if (p < hMatProp[mat][4]) p = hMatProp[mat][4];

//    c = hMatProp[mat][1];

    return p;
}


float pressureTaitHost(int mat, float rho, float u) {
    /**
     * \brief Tait Equation Of State
     *
     * p = rho0 * c0 * c0 / 7.0 * (powf((rho / rho0), 7) - 1.0);
     * c = c0;
     *
     * rho0 = dMatProp[mat][0];
     * c0 = dMatProp[mat][1];
     * pmin = dMatProp[mat][2];
     *
     * \date Jun 10, 2010
     * \author Luca Massidda
     */

    float p;
//    float c;

    p = hMatProp[mat][0] * powf(hMatProp[mat][1], 2) / 7.0
        * (powf((rho / hMatProp[mat][0]), 7) - 1.0);

    if (p < hMatProp[mat][2]) p = hMatProp[mat][2];

//    c = hMatProp[mat][1];

    return p;
}


// Global code

__global__ void balanceMassMomentumDevice(const int* dList,
	const float* dPosX, const float* dPosY,
	const float* dVelX, const float* dVelY, 
	const float* dDensity, const float* dPressure, 
	float* dDensityDot, float* dVelDotX, float* dVelDotY) {
	
    /**
     * \brief Interate particles
     *
     * \date Jan 6, 2011
     * \author Luca Massidda
     */

	int ip, il, jp;
    float iDensityDot;
    float iVelDotX, iVelDotY;
    volatile float dx, dy, dz, dr, dvr, dwdr, f;
	
    ip = threadIdx.x + blockDim.x * blockIdx.x;
	
    if (ip < dPN) {
		iDensityDot = 0.0;
		iVelDotX = 0.0;
		iVelDotY = 0.0;
		
		for (il = 0; il < MAXN; il++) {
			jp = dList[ip * MAXN + il];
			
			dx = dPosX[ip] - dPosX[jp];
			dy = dPosY[ip] - dPosY[jp];
			dz = 0.0;
			dr = sqrtf(dx * dx + dy * dy + dz * dz);
			
			if (dr < 0.1 * dSmooth) dr = 100.0 * dSmooth;
			
			//dwdr = kernelDerivGauss(dr, dSmooth);
			dwdr = kernelDerivWendland(dr, dSmooth);
			
			dvr = 0.0;
			dvr += (dPosX[ip] - dPosX[jp]) * (dVelX[ip] - dVelX[jp]);
			dvr += (dPosY[ip] - dPosY[jp]) * (dVelY[ip] - dVelY[jp]);
			
			iDensityDot += dMass * dvr * dwdr / dr;
			
			// Calculate interparticle pressure action
			f = -(dPressure[ip] + dPressure[jp])
				/ (dDensity[ip] * dDensity[jp]);
			
			iVelDotX += dMass * f * dwdr * (dPosX[ip] - dPosX[jp]) / dr;
			iVelDotY += dMass * f * dwdr * (dPosY[ip] - dPosY[jp]) / dr;
			
			// Calculate shock correction for mass
			f = dDensity[ip] - dDensity[jp];
			f *= 2.0 * dSound / (dDensity[ip] + dDensity[jp]);
			
			iDensityDot += dMass * f * dwdr;
			
			// Calculate shock correction for momentum
			if (dvr < 0) f = dvr;
			else f = 0.0;
			
			f *= dSmooth / (dr * dr + 0.01 * dSmooth * dSmooth);
			f *= 2. * dSound / (dDensity[ip] + dDensity[jp]);
			f *= 0.03;
			
			iVelDotX += dMass * f * dwdr * (dPosX[ip] - dPosX[jp]) / dr;
			iVelDotY += dMass * f * dwdr * (dPosY[ip] - dPosY[jp]) / dr;
		}
		
		dDensityDot[ip] += iDensityDot;
		dVelDotX[ip] += iVelDotX;
		dVelDotY[ip] += iVelDotY;

    }
}


void balanceMassMomentumHost(void) {
	
    /**
     * \brief Interate particles
     *
     * \date Jan 6, 2011
     * \author Luca Massidda
     */

	int ip, il, jp;
    float iDensityDot;
    float iVelDotX, iVelDotY;
    float dx, dy, dz, dr, dvr, dwdr, f;
	
    for (ip = 0; ip < hPN; ip++) {
		iDensityDot = 0.0;
		iVelDotX = 0.0;
		iVelDotY = 0.0;
		
		for (il = 0; il < MAXN; il++) {
			jp = hList[ip * MAXN + il];
			
			dx = hPosX[ip] - hPosX[jp];
			dy = hPosY[ip] - hPosY[jp];
			dz = 0.0;
			dr = sqrtf(dx * dx + dy * dy + dz * dz);
			
			if (dr < 0.1 * hSmooth) dr = 100.0 * hSmooth;
			
			//dwdr = kernelDerivGauss(dr, hSmooth);
			dwdr = kernelDerivWendlandHost(dr, hSmooth);
			
			dvr = 0.0;
			dvr += (hPosX[ip] - hPosX[jp]) * (hVelX[ip] - hVelX[jp]);
			dvr += (hPosY[ip] - hPosY[jp]) * (hVelY[ip] - hVelY[jp]);
			
			iDensityDot += hMass * dvr * dwdr / dr;
			
			// Calculate interparticle pressure action
			f = -(hPressure[ip] + hPressure[jp])
				/ (hDensity[ip] * hDensity[jp]);
			
			iVelDotX += hMass * f * dwdr * (hPosX[ip] - hPosX[jp]) / dr;
			iVelDotY += hMass * f * dwdr * (hPosY[ip] - hPosY[jp]) / dr;
			
			// Calculate shock correction for mass
			f = hDensity[ip] - hDensity[jp];
			f *= 2.0 * hSound / (hDensity[ip] + hDensity[jp]);
			
			iDensityDot += hMass * f * dwdr;
			
			// Calculate shock correction for momentum
			if (dvr < 0) f = dvr;
			else f = 0.0;
			
			f *= hSmooth / (dr * dr + 0.01 * hSmooth * hSmooth);
			f *= 2. * hSound / (hDensity[ip] + hDensity[jp]);
			f *= 0.03;
			
			iVelDotX += hMass * f * dwdr * (hPosX[ip] - hPosX[jp]) / dr;
			iVelDotY += hMass * f * dwdr * (hPosY[ip] - hPosY[jp]) / dr;
		}
		
		hDensityDot[ip] += iDensityDot;
		hVelDotX[ip] += iVelDotX;
		hVelDotY[ip] += iVelDotY;

    }
}

__global__ void balanceEnergyDevice(const float* dPressure,
		const float* dDensity, const float* dDensityDot,
		float* dEnergyDot) {

    /**
     * \brief Interate particles
     *
     * \date Jan 9, 2011
     * \author Luca Massidda
     */

    volatile int ip;
    float iPressure, iDensity, iDensityDot;
    float iEnergyDot;

    ip = threadIdx.x + blockDim.x * blockIdx.x;

    if (ip < dPN) {
        iPressure = dPressure[ip];
        iDensity = dDensity[ip];
        iDensityDot = dDensityDot[ip];

        iEnergyDot = (iPressure * iDensityDot) / (iDensity * iDensity);

        dEnergyDot[ip] += iEnergyDot;
    }
}


void balanceEnergyHost(void) {

    /**
     * \brief Interate particles
     *
     * \date Jan 9, 2011
     * \author Luca Massidda
     */

    int ip;
    float iPressure, iDensity, iDensityDot;
    float iEnergyDot;

    for (ip = 0; ip < hPN; ip++) {
        iPressure = hPressure[ip];
        iDensity = hDensity[ip];
        iDensityDot = hDensityDot[ip];

        iEnergyDot = (iPressure * iDensityDot) / (iDensity * iDensity);

        hEnergyDot[ip] += iEnergyDot;
    }
}


__global__ void updateParticlesDevice(const int* dMaterial,
		const float* dVelDotX, const float* dVelDotY,
		const float* dDensityDot, const float* dEnergyDot, const float alpha, 
		const float* dPosX0, const float* dPosY0,
		const float* dVelX0, const float* dVelY0,
		const float* dDensity0, const float* dEnergy0,
		float* dPosX, float* dPosY,
		float* dVelX, float* dVelY, 
		float* dDensity, float* dEnergy, float* dPressure) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    int ip;
    float f;
    int iMaterial;
    float iDensity, iEnergy;

    ip = threadIdx.x + blockDim.x * blockIdx.x;

    if (ip < dPN) {
        f = dPosX0[ip] + alpha * (dPosX[ip] + dRun.dt * dVelX[ip] - dPosX0[ip]);
        /*
		if (f < dRun.minX)
			f += dRun.maxX - dRun.minX;
		if (f > dRun.maxX)
			f -= dRun.maxX - dRun.minX;
		*/
        dPosX[ip] = f;

        f = dPosY0[ip] + alpha * (dPosY[ip] + dRun.dt * dVelY[ip] - dPosY0[ip]);
        /*
		if (f < dRun.minY)
			f += dRun.maxY - dRun.minY;
		if (f > dRun.maxY)
			f -= dRun.maxY - dRun.minY;
		*/
        dPosY[ip] = f;

        f = dVelX0[ip] + alpha * (dVelX[ip] + dRun.dt * dVelDotX[ip] - dVelX0[ip]);
        dVelX[ip] = f;
        
        f = dVelY0[ip] + alpha * (dVelY[ip] + dRun.dt * dVelDotY[ip] - dVelY0[ip]);
        dVelY[ip] = f;

        f = dDensity0[ip] + alpha * (dDensity[ip] + dRun.dt * dDensityDot[ip] - dDensity0[ip]);
        dDensity[ip] = f;

        f = dEnergy0[ip] + alpha * (dEnergy[ip] + dRun.dt * dEnergyDot[ip] - dEnergy0[ip]);
        dEnergy[ip] = f;
        
        iMaterial = dMaterial[ip];
        
        if (iMaterial < 0) {
			dVelX[ip] = dVelX0[ip];
			dVelY[ip] = dVelY0[ip];
        }

        iMaterial = abs(iMaterial);
        iDensity = dDensity[ip];
        iEnergy = dEnergy[ip];

        switch (dMatType[iMaterial]) {
        case (1) : // IDEAL GAS EOS
            dPressure[ip] = pressureGas(iMaterial, iDensity, iEnergy);
            break;
        case (2) : // MIE-GRUNEISEN POLYNOMIAL EOS
            dPressure[ip] = pressurePoly(iMaterial, iDensity, iEnergy);
            break;
        case (3) : // MIE-GRUNEISEN SHOCK EOS
            dPressure[ip] = pressureShock(iMaterial, iDensity, iEnergy);
            break;
        case (4) : // TAIT EOS
            dPressure[ip] = pressureTait(iMaterial, iDensity, iEnergy);
            break;
        default :
            dPressure[ip] = 0.0;
        }
        
        
	}
}


void updateParticlesHost(const float alpha) {

    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

    int ip;
    float f;
    int iMaterial;
    float iDensity, iEnergy;

    for (ip = 0; ip < hPN; ip++) {
        f = hPosX0[ip] + alpha * (hPosX[ip] + hRun.dt * hVelX[ip] - hPosX0[ip]);
        /*
		if (f < hRun.minX)
			f += hRun.maxX - hRun.minX;
		if (f > hRun.maxX)
			f -= hRun.maxX - hRun.minX;
		*/
        hPosX[ip] = f;

        f = hPosY0[ip] + alpha * (hPosY[ip] + hRun.dt * hVelY[ip] - hPosY0[ip]);
        /*
		if (f < hRun.minY)
			f += hRun.maxY - hRun.minY;
		if (f > hRun.maxY)
			f -= hRun.maxY - hRun.minY;
		*/
        hPosY[ip] = f;

        f = hVelX0[ip] + alpha * (hVelX[ip] + hRun.dt * hVelDotX[ip] - hVelX0[ip]);
        hVelX[ip] = f;
        
        f = hVelY0[ip] + alpha * (hVelY[ip] + hRun.dt * hVelDotY[ip] - hVelY0[ip]);
        hVelY[ip] = f;

        f = hDensity0[ip] + alpha * (hDensity[ip] + hRun.dt * hDensityDot[ip] - hDensity0[ip]);
        hDensity[ip] = f;

        f = hEnergy0[ip] + alpha * (hEnergy[ip] + hRun.dt * hEnergyDot[ip] - hEnergy0[ip]);
        hEnergy[ip] = f;
        
        iMaterial = hMaterial[ip];
        
        if (iMaterial < 0) {
			hVelX[ip] = hVelX0[ip];
			hVelY[ip] = hVelY0[ip];
        }

        iMaterial = abs(iMaterial);
        iDensity = hDensity[ip];
        iEnergy = hEnergy[ip];

        switch (hMatType[iMaterial]) {
        case (1) : // IDEAL GAS EOS
            hPressure[ip] = pressureGasHost(iMaterial, iDensity, iEnergy);
            break;
        case (2) : // MIE-GRUNEISEN POLYNOMIAL EOS
            hPressure[ip] = pressurePolyHost(iMaterial, iDensity, iEnergy);
            break;
        case (3) : // MIE-GRUNEISEN SHOCK EOS
            hPressure[ip] = pressureShockHost(iMaterial, iDensity, iEnergy);
            break;
        case (4) : // TAIT EOS
            hPressure[ip] = pressureTaitHost(iMaterial, iDensity, iEnergy);
            break;
        default :
            hPressure[ip] = 0.0;
        }
        
        
	}
}


__global__ void updateForcesDevice(const int* dMaterial,
		float* dVelDotX, float* dVelDotY, float* dDensityDot,
		float* dEnergyDot) {
	
    int ip;
    int iMaterial;
    float iVelDotX, iVelDotY, iDensityDot, iEnergyDot;

    ip = threadIdx.x + blockDim.x * blockIdx.x;

    if (ip < dPN) {
        iVelDotX = 0.0;
        iVelDotY = 0.0;
        iDensityDot = 0.0;
        iEnergyDot = 0.0;

        iMaterial = dMaterial[ip];

        if (iMaterial > 0) iVelDotY = -9.81;

        dVelDotX[ip] = iVelDotX;
        dVelDotY[ip] = iVelDotY;
        dDensityDot[ip] = iDensityDot;
        dEnergyDot[ip] = iEnergyDot;
    }
}

void updateForcesHost(void) {
	
    int ip;
    int iMaterial;
    float iVelDotX, iVelDotY, iDensityDot, iEnergyDot;

    for (ip = 0; ip < hPN; ip++) {
        iVelDotX = 0.0;
        iVelDotY = 0.0;
        iDensityDot = 0.0;
        iEnergyDot = 0.0;

        iMaterial = hMaterial[ip];

        if (iMaterial > 0) iVelDotY = -9.81;

        hVelDotX[ip] = iVelDotX;
        hVelDotY[ip] = iVelDotY;
        hDensityDot[ip] = iDensityDot;
        hEnergyDot[ip] = iEnergyDot;
    }
}


__global__ void kerSortFloat(float* dArrayOut, const float* dArrayIn, 
	const int* dIndex) {
	
    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

	int ip;
	
	ip = threadIdx.x + blockDim.x * blockIdx.x;
	if (ip >= dPN) return;
	
	dArrayOut[ip] = dArrayIn[dIndex[ip]];
}


__global__ void kerSortInt(int* dArrayOut, const int* dArrayIn, 
	const int* dIndex) {
	
    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

	int ip;
	
	ip = threadIdx.x + blockDim.x * blockIdx.x;
	if (ip >= dPN) return;
	
	dArrayOut[ip] = dArrayIn[dIndex[ip]];
}


__global__ void updateHashDevice(const struct grid dGrid, 
	const float* dPosX, const float* dPosY, 
	int* dHash, int* dIndex) {
	
    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

	int ip, ix, iy, ic;
	
    ip = threadIdx.x + blockDim.x * blockIdx.x;
	
    if (ip < dPN) {
        ix = (int) truncf((dPosX[ip] - dGrid.oX) / dGrid.size);
        iy = (int) truncf((dPosY[ip] - dGrid.oY) / dGrid.size);
		ic = ix + iy * dGrid.nX;
		
		dHash[ip] = ic;
		dIndex[ip] = ip;
	}
}


__global__ void updateSetsDevice(int *dSetStart, int *dSetStop, 
	const int* dHash) {
	
    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */
	
	__shared__ int prevHash[THREADS];
	__shared__ int nextHash[THREADS];
	
	int ip;
	int hash;
	
    ip = threadIdx.x + blockDim.x * blockIdx.x;
    if (ip >= dPN) return;
	
	hash = dHash[ip];
	
	if (threadIdx.x < THREADS -1) prevHash[threadIdx.x +1] = hash;
	if (threadIdx.x == 0) {
		if (ip == 0) prevHash[threadIdx.x] = -1;
		else prevHash[threadIdx.x] = dHash[ip -1];
	}
	
	if (threadIdx.x > 0) nextHash[threadIdx.x -1] = hash;
	if (threadIdx.x == THREADS -1) {
		if (ip == dPN -1) nextHash[threadIdx.x] = -1;
		else nextHash[threadIdx.x] = dHash[ip +1];
	}
	
	__syncthreads();
	
	if (hash != prevHash[threadIdx.x]) dSetStart[hash] = ip;
	
	if (hash != nextHash[threadIdx.x]) dSetStop[hash] = ip +1;
	
}


__global__ void updateListDevice(int *dList, 
	const int* dSetStart, const int* dSetStop, 
	const struct grid dGrid, 
	const float* dPosX, const float* dPosY) {
	
	int ip, ic, ix, iy, i, j, jp, jc, np;
	float dx, dy, dr;
	
    // Particles list is filled
    ip = threadIdx.x + blockDim.x * blockIdx.x;
	if (ip >= dPN) return;
	
	ix = (int) ((dPosX[ip] - dGrid.oX) / dGrid.size);
	iy = (int) ((dPosY[ip] - dGrid.oY) / dGrid.size);
	ic = ix + iy * dGrid.nX;
	np = 0;
	
	for (j = -1; j <= 1; j++) {
		for (i = -1; i <= 1; i++) {
			jc = ic + i + j * dGrid.nX;
			
			for (jp = dSetStart[jc]; jp < dSetStop[jc]; jp++) {
				dx = dPosX[ip] - dPosX[jp];
				dy = dPosY[ip] - dPosY[jp];
				dr = sqrtf(dx * dx + dy * dy);
				
				if ((dr < 2.0 * dSmooth) && (np < MAXN)) {
					dList[ip * MAXN + np] = jp;
					np++;
				}
					
			}
				
		}
	}
	
	while (np < MAXN) {
		dList[ip * MAXN + np] = ip;
		np++;
	}
	
}



// Host code

int initHost() {

    hMaterial = (int *) malloc(MAXP * sizeof(int));
    hPosX = (float *) malloc(MAXP * sizeof(float));
    hPosY = (float *) malloc(MAXP * sizeof(float));
    hVelX = (float *) malloc(MAXP * sizeof(float));
    hVelY = (float *) malloc(MAXP * sizeof(float));
    hDensity = (float *) malloc(MAXP * sizeof(float));
    hEnergy = (float *) malloc(MAXP * sizeof(float));
    hPressure = (float *) malloc(MAXP * sizeof(float));
    hVelDotX = (float *) malloc(MAXP * sizeof(float));
    hVelDotY = (float *) malloc(MAXP * sizeof(float));
    hDensityDot = (float *) malloc(MAXP * sizeof(float));
    hEnergyDot = (float *) malloc(MAXP * sizeof(float));
	
    hList = (int *) malloc(MAXP * MAXN * sizeof(int));
    hHash = (int *) malloc(MAXP * sizeof(int));
    hIndex = (int *) malloc(MAXP * sizeof(int));
    hSetStart = (int *) malloc(MAXG * sizeof(int));
    hSetStop = (int *) malloc(MAXG * sizeof(int));
	
    hGrid.set = (int *) malloc(MAXG * sizeof(int));
    hGrid.nump = (int *) malloc(MAXG * sizeof(int));
    hGrid.cell = (int *) malloc(MAXG * sizeof(int));
    hGrid.perm = (int *) malloc(MAXP * sizeof(int));

    hPosX0 = (float *) malloc(MAXP * sizeof(float));
    hPosY0 = (float *) malloc(MAXP * sizeof(float));
    hVelX0 = (float *) malloc(MAXP * sizeof(float));
    hVelY0 = (float *) malloc(MAXP * sizeof(float));
    hDensity0 = (float *) malloc(MAXP * sizeof(float));
    hEnergy0 = (float *) malloc(MAXP * sizeof(float));
    
    return 0;
}


int initDevice() {

    cutilSafeCall( hipMalloc((void**) &(dMaterial), (MAXP * sizeof(int))) );
    cutilSafeCall( hipMalloc((void**) &(dPosX), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dPosY), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dVelX), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dVelY), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dDensity), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dEnergy), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dPressure), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dVelDotX), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dVelDotY), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dDensityDot), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dEnergyDot), (MAXP * sizeof(float))) );
    
    cutilSafeCall( hipMalloc((void**) &(dList), (MAXP * MAXN * sizeof(int))) );
    cutilSafeCall( hipMalloc((void**) &(dHash), (MAXP * sizeof(int))) );
    cutilSafeCall( hipMalloc((void**) &(dIndex), (MAXP * sizeof(int))) );
    
    cutilSafeCall( hipMalloc((void**) &(dSetStart), (MAXG * sizeof(int))) );
    cutilSafeCall( hipMalloc((void**) &(dSetStop), (MAXG * sizeof(int))) );
    
    cutilSafeCall( hipMalloc((void**) &(dIntDummy), (MAXP * sizeof(int))) );
    cutilSafeCall( hipMalloc((void**) &(dFloatDummy), (MAXP * sizeof(float))) );

    dGrid.oX = hGrid.oX;
    dGrid.oY = hGrid.oY;
    dGrid.nX = hGrid.nX;
    dGrid.nY = hGrid.nY;
    dGrid.size = hGrid.size;
    dGrid.SN = hGrid.SN;
    cutilSafeCall( hipMalloc((void**) &(dGrid.set), (MAXG * sizeof(int))) );
    cutilSafeCall( hipMalloc((void**) &(dGrid.nump), (MAXG * sizeof(int))) );
    cutilSafeCall( hipMalloc((void**) &(dGrid.cell), (MAXG * sizeof(int))) );
    cutilSafeCall( hipMalloc((void**) &(dGrid.perm), (MAXP * sizeof(int))) );

    cutilSafeCall( hipMalloc((void**) &(dPosX0), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dPosY0), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dVelX0), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dVelY0), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dDensity0), (MAXP * sizeof(float))) );
    cutilSafeCall( hipMalloc((void**) &(dEnergy0), (MAXP * sizeof(float))) );

    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("dPN"), &hPN, sizeof(int)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("dSmooth"), &hSmooth, sizeof(float)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("dMass"), &hMass, sizeof(float)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("dSound"), &hSound, sizeof(float)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("dRun"), &hRun, sizeof(struct simulation)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("dMatType"), hMatType, 10 * sizeof(int)) );
    cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL("dMatProp"), hMatProp, 100 * sizeof(float)) );

    return 0;
}


int initCUDPP() {

    CUDPPConfiguration sortConfig;
    sortConfig.algorithm = CUDPP_SORT_RADIX;
    sortConfig.datatype = CUDPP_UINT;
    sortConfig.op = CUDPP_ADD;
    sortConfig.options = CUDPP_OPTION_KEY_VALUE_PAIRS;
    cudppPlan(&hSortHandle, sortConfig, hPN, 1, 0);
    
    return 0;
}


int copyHostToDevice() {

    cutilSafeCall( hipMemcpy(dMaterial, hMaterial,
                              (MAXP * sizeof(int)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dPosX, hPosX,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dPosY, hPosY,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dVelX, hVelX,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dVelY, hVelY,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dDensity, hDensity,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dEnergy, hEnergy,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dPressure, hPressure,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dVelDotX, hVelDotX,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dVelDotY, hVelDotY,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dDensityDot, hDensityDot,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dEnergyDot, hEnergyDot,
                              (MAXP * sizeof(float)), hipMemcpyHostToDevice) );

    cutilSafeCall( hipMemcpy(dList, hList,
                              (MAXP * MAXN * sizeof(int)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dHash, hHash,
                              (MAXP * sizeof(int)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dIndex, hIndex,
                              (MAXP * sizeof(int)), hipMemcpyHostToDevice) );
                              
    cutilSafeCall( hipMemcpy(dSetStart, hSetStart,
                              (MAXG * sizeof(int)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dSetStop, hSetStop,
                              (MAXG * sizeof(int)), hipMemcpyHostToDevice) );
	
    dGrid.oX = hGrid.oX;
    dGrid.oY = hGrid.oY;
    dGrid.nX = hGrid.nX;
    dGrid.nY = hGrid.nY;
    dGrid.size = hGrid.size;
    dGrid.SN = hGrid.SN;
    cutilSafeCall( hipMemcpy(dGrid.set, hGrid.set,
                              (MAXG * sizeof(int)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dGrid.nump, hGrid.nump,
                              (MAXG * sizeof(int)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dGrid.cell, hGrid.cell,
                              (MAXG * sizeof(int)), hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(dGrid.perm, hGrid.perm,
                              (MAXP * sizeof(int)), hipMemcpyHostToDevice) );

    return 0;
}


int copyDeviceToHost() {

    cutilSafeCall( hipMemcpy(hMaterial, dMaterial,
                              (MAXP * sizeof(int)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hPosX, dPosX,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hPosY, dPosY,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hVelX, dVelX,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hVelY, dVelY,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hDensity, dDensity,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hEnergy, dEnergy,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hPressure, dPressure,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hVelDotX, dVelDotX,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hVelDotY, dVelDotY,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hDensityDot, dDensityDot,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hEnergyDot, dEnergyDot,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToHost) );

    cutilSafeCall( hipMemcpy(hList, dList,
                              (MAXP * MAXN * sizeof(int)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hHash, dHash,
                              (MAXP * sizeof(int)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hIndex, dIndex,
                              (MAXP * sizeof(int)), hipMemcpyDeviceToHost) );
                              
    cutilSafeCall( hipMemcpy(hSetStart, dSetStart,
                              (MAXG * sizeof(int)), hipMemcpyDeviceToHost) );
    cutilSafeCall( hipMemcpy(hSetStop, dSetStop,
                              (MAXG * sizeof(int)), hipMemcpyDeviceToHost) );
	
    return 0;
}

int backupData() {

    cutilSafeCall( hipMemcpy(dPosX0, dPosX,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
    cutilSafeCall( hipMemcpy(dPosY0, dPosY,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
    cutilSafeCall( hipMemcpy(dVelX0, dVelX,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
    cutilSafeCall( hipMemcpy(dVelY0, dVelY,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
    cutilSafeCall( hipMemcpy(dDensity0, dDensity,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
    cutilSafeCall( hipMemcpy(dEnergy0, dEnergy,
                              (MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );

    return 0;
}

int backupDataHost() {

    memcpy(hPosX0, hPosX, MAXP * sizeof(float));
    memcpy(hPosY0, hPosY, MAXP * sizeof(float));
    memcpy(hVelX0, hVelX, MAXP * sizeof(float));
    memcpy(hVelY0, hVelY, MAXP * sizeof(float));
    memcpy(hDensity0, hDensity, MAXP * sizeof(float));
    memcpy(hEnergy0, hEnergy, MAXP * sizeof(float));

    return 0;
}


int initRun() {

    /**
     * \brief Input run data
     *
     * Reads the input file for run data
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    char tok[10];
    int i, m, p, pn;
    int iv;
    float fv;
    int mpn, mpp[10];

    // Open stream file
    stream = fopen("armando.run", "r");

    while (!feof(stream)) {
        sprintf(tok, " ");
        fscanf(stream, "%s", tok);

        if (strcmp(tok, "MAT") == 0) {
            fscanf(stream, "%i", &iv);
            if ((iv > 0) && (iv <= 50))
                m = iv;

            for (p = 0; p < 10; p++)
                hMatProp[m][p] = 0.0;

            if ((m > 0) && (m <= 10))
                pn = 3;
            if ((m > 10) && (m <= 20))
                pn = 9;
            if ((m > 20) && (m <= 30))
                pn = 10;
            if ((m > 30) && (m <= 40))
                pn = 5;
            if ((m > 40) && (m <= 50))
                pn = 3;

            for (p = 0; p < pn; p++) {
                fscanf(stream, "%f", &fv);
                hMatProp[m][p] = fv;
            }

            printf("Material %d\n", m);
            printf("hMatProp: \n");
            for (p = 0; p < pn; p++)
                printf(" %f\n", hMatProp[m][p]);
            printf("\n");
        }

        if (strcmp(tok, "TIME") == 0) {
            fscanf(stream, "%f", &fv);
            if (fv > 0.0)
                hRun.dt = fv;

            fscanf(stream, "%i", &iv);
            if (iv > 0)
                hRun.tsn = iv;

            fscanf(stream, "%i", &iv);
            if (iv > 0)
                hRun.ssi = iv;

            printf("Time step: %f\n", hRun.dt);
            printf("Steps: %i\n", hRun.tsn);
            printf("Save step: %i\n", hRun.ssi);
            printf("\n");
        }

        if (strcmp(tok, "LIMITS") == 0) {
            fscanf(stream, "%f", &fv);
            hRun.minX = fv;

            fscanf(stream, "%f", &fv);
            hRun.maxX = fv;

            fscanf(stream, "%f", &fv);
            hRun.minY = fv;

            fscanf(stream, "%f", &fv);
            hRun.maxY = fv;

            printf("Domain limits: \n");
            printf("X: %+e - %+e \n", hRun.minX, hRun.maxX);
            printf("Y: %+e - %+e \n", hRun.minY, hRun.maxY);
            printf("\n");
        }

        if (strcmp(tok, "MONITORS") == 0) {
            fscanf(stream, "%i", &iv);
            mpn = iv;

            for (i = 0; i < mpn; i++) {
                fscanf(stream, "%i", &iv);
                mpp[i] = iv;
            }

            printf("Monitored particles: %i \n", mpn);
            if (mpn > 0) {
                printf("Index:");
                for (i = 0; i < mpn; i++)
                    printf(" %i", mpp[i]);
                printf("\n");
                printf("\n");
            }
        }
    }

    fclose(stream);

    hSound = hSmooth / hRun.dt;

    return 0;
}

int scanData() {
    /**
     * \brief Input particle data file
     *
     * Reads particle data from a disk file
     *
     * \date Oct 20, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    int i;
    float fv1, fv2, fv3;
    int iv;

    // Stream file position
    stream = fopen("in_pos.txt", "r");
    for (i = 0; !feof(stream); i++) {
        fscanf(stream, "%e %e ", &fv1, &fv2);
        hPosX[i] = fv1;
        hPosY[i] = fv2;
    }
    fclose(stream);
    hPN = i;

    // Stream file velocity
    stream = fopen("in_vel.txt", "r");
    for (i = 0; i < hPN; i++) {
        fscanf(stream, "%e %e", &fv1, &fv2);
        hVelX[i] = fv1;
        hVelY[i] = fv2;
    }
    fclose(stream);

    // Stream file info
    stream = fopen("in_info.txt", "r");
    for (i = 0; i < hPN; i++) {
        fscanf(stream, "%i %e %e ", &iv, &fv1, &fv2);
        hMaterial[i] = iv;
        hMass = fv1;
        hSmooth = fv2;
    }
    fclose(stream);

    // Stream file field
    stream = fopen("in_field.txt", "r");
    for (i = 0; i < hPN; i++) {
        fscanf(stream, "%e %e %e ", &fv1, &fv2, &fv3);
        hDensity[i] = fv1;
        hPressure[i] = fv2;
        hEnergy[i] = fv3;
    }
    fclose(stream);

    return 0;
}

int printData() {
    /**
     * \brief Particle data file output
     *
     * Saves particle data on a disk file
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    int i;

    // Stream file position
    stream = fopen("new_pos.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e\n", hPosX[i], hPosY[i]);
    fclose(stream);

    // Stream file velocity
    stream = fopen("new_vel.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e \n", hVelX[i], hVelY[i]);
    fclose(stream);

    // Stream file info
    stream = fopen("new_info.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%i %+14.8e %+14.8e \n", hMaterial[i], hMass, hSmooth);
    fclose(stream);

    // Stream file field
    stream = fopen("new_field.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e \n", hDensity[i], hPressure[i], hEnergy[i]);
    fclose(stream);

    // Stream file add1
    stream = fopen("new_debug.txt", "w");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e %+14.8e \n", hDensityDot[i],
                hVelDotX[i], hVelDotY[i], hEnergyDot[i]);
    fclose(stream);

    return 0;
}


int outputCase() {
    /**
     * \brief Output Case file
     *
     * Saves ensight case file
     *
     * \date Jul 5, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    int ts;

    // Open stream file
    stream = fopen("armando.case", "w");

    fprintf(stream, "# Ensight formatted case file for Armando\n");
    fprintf(stream, "\n");
    fprintf(stream, "FORMAT\n");
    fprintf(stream, "type: ensight gold\n");
    fprintf(stream, "\n");
    fprintf(stream, "GEOMETRY\n");
    fprintf(stream, "model:    1           armando_pos_*****.geo\n");
    fprintf(stream, "\n");
    fprintf(stream, "VARIABLE\n");
    fprintf(stream, "vector per node:    1 velocity armando_vel_*****.dat\n");
    fprintf(stream, "scalar per node:    1 density  armando_rho_*****.dat\n");
    fprintf(stream, "scalar per node:    1 pressure armando_pre_*****.dat\n");
    fprintf(stream, "scalar per node:    1 energy   armando_ene_*****.dat\n");
    fprintf(stream, "\n");
    fprintf(stream, "TIME\n");
    fprintf(stream, "time set: %i\n", 1);
    fprintf(stream, "number of steps: %i\n", (hRun.tsn / hRun.ssi + 1));
    fprintf(stream, "filename start number: %i\n", 0);
    fprintf(stream, "filename increment: %i\n", 1);
    fprintf(stream, "time values:\n");

    for (ts = 0; ts <= hRun.tsn; ts++)
        if ((ts % hRun.ssi) == 0)
            fprintf(stream, "%14.8e\n", (ts * hRun.dt));

    // Close stream file
    fclose(stream);

    return 0;
}

int outputData(int ss) {
    /**
     * \brief Output Data file
     *
     * Saves ensight data file
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    char filename[80];
    int i;

    // Stream position file
    sprintf(filename, "armando_pos_%05d.geo", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "Armando output in EnSight Gold format\n");
    fprintf(stream, "EnSight 8.0.7\n");
    fprintf(stream, "node id assign\n");
    fprintf(stream, "element id assign\n");
    fprintf(stream, "extents\n");
    fprintf(stream, " 1.00000e+38-1.00000e+38\n");
    fprintf(stream, " 1.00000e+38-1.00000e+38\n");
    fprintf(stream, " 1.00000e+38-1.00000e+38\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "SPH particles\n");
    fprintf(stream, "coordinates\n");
    fprintf(stream, "%10i\n", hPN);

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hPosX[i]);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hPosY[i]);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", 0.0);

    fclose(stream);

    // Stream velocity file
    sprintf(filename, "armando_vel_%05d.dat", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "particle velocity in EnSight Gold format\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "coordinates\n");

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hVelX[i]);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hVelY[i]);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", 0.0);

    fclose(stream);

    // Stream density file
    sprintf(filename, "armando_rho_%05d.dat", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "particle density in EnSight Gold format\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "coordinates\n");

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hDensity[i]);

    fclose(stream);

    // Stream pressure file
    sprintf(filename, "armando_pre_%05d.dat", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "particle pressure in EnSight Gold format\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "coordinates\n");

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hPressure[i]);

    fclose(stream);

    // Stream energy file
    sprintf(filename, "armando_ene_%05d.dat", ss);
    stream = fopen(filename, "w");

    fprintf(stream, "particle energy in EnSight Gold format\n");
    fprintf(stream, "part\n");
    fprintf(stream, "%10i\n", 1);
    fprintf(stream, "coordinates\n");

    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e\n", hEnergy[i]);

    fclose(stream);

    return 0;
}


int outputVTK(int ss) {
    /**
     * \brief Output Data file
     *
     * Saves vtk data file
     *
     * \date Oct 21, 2010
     * \author Luca Massidda
     */

    FILE *stream;
    char filename[80];
    int i;

    // Stream position file
    sprintf(filename, "out%05d.vtk", ss);
    stream = fopen(filename, "w");
	
	fprintf(stream, "# vtk DataFile Version 2.0\n");
    fprintf(stream, "Unstructured Grid Example\n");
    fprintf(stream, "ASCII\n");
    fprintf(stream, "DATASET UNSTRUCTURED_GRID\n");

    fprintf(stream, "POINTS %i float\n", hPN);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e %+e %+e \n", hPosX[i], hPosY[i], 0.0);

    fprintf(stream, "CELLS %i %i \n", hPN, 2*hPN);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%i %i \n", 1, i);

    fprintf(stream, "CELL_TYPES %i \n", hPN);
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%i \n", 1);
    
    fprintf(stream, "POINT_DATA %i \n", hPN);
    
    fprintf(stream, "SCALARS density float 1 \n", hPN);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e \n", hDensity[i]);
    
    fprintf(stream, "SCALARS pressure float 1 \n", hPN);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e \n", hPressure[i]);
    
    fprintf(stream, "SCALARS energy float 1 \n", hPN);
    fprintf(stream, "LOOKUP_TABLE default\n");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e \n", hEnergy[i]);
    
    fprintf(stream, "VECTORS velocity float\n");
    for (i = 0; i < hPN; i++)
        fprintf(stream, "%+e %+e %+e \n", hVelX[i], hVelY[i], 0.0);
    
    fclose(stream);

    return 0;
}


void initDamBreak() {

    int i, j, m, pi;
    double rho, c0, pmin;
    double dr;

    m = 1;
    rho = 1000.;
    c0 = 50.;
    pmin = -1.e12;

    hMatType[m] = 4;
    hMatProp[m][0] = rho;
    hMatProp[m][1] = c0;
    hMatProp[m][2] = pmin;

    dr = 0.02 / 4; // x4
    pi = 0;

    for (j = 0; j <= 50 * 4 ; j++) {
        for (i = 0; i <= 100 * 4; i++) {
            hPosX[pi] = i * dr + 0.5 * dr;
            hPosY[pi] = j * dr + 0.5 * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = m;
            hDensity[pi] = rho; //+ (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
            pi++;
        }
    }
    // 0 - 268   0 - 150
    /*
        for (j = 151; j <= 153; j++) {
            for (i = -3; i <= 271; i++) {
                hPosX[pi] = i * dr;
                hPosY[pi] = j * dr;

                hVelX[pi] = 0.0;
                hVelY[pi] = 0.0;
                hMaterial[pi] = -m;
                hDensity[pi] = rho; // + (9.81 * rho / c0 / c0 * (50 - j) * dr);
                hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
                pi++;
            }
        }
    */
    for (j = -3; j <= -1; j++) {
        for (i = -3; i <= 269 * 4 + 2; i++) {
            hPosX[pi] = i * dr;
            hPosY[pi] = j * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = -m;
            hDensity[pi] = rho; // + (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
            pi++;
        }
    }

    for (j = -0; j <= 80 * 4; j++) {
        for (i = -3; i <= -1; i++) {
            hPosX[pi] = i * dr;
            hPosY[pi] = j * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = -m;
            hDensity[pi] = rho; // + (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
            pi++;
        }
    }

    for (j = -0; j <= 80 * 4; j++) {
        for (i = 269 * 4; i <= 269 * 4 +2; i++) {
            hPosX[pi] = i * dr;
            hPosY[pi] = j * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = -m;
            hDensity[pi] = rho; // + (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 0.0;
            pi++;
        }
    }

    hPN = pi;
    hSmooth = 1.2 * dr;
    hMass = rho * dr * dr;
    hSound = c0;

    hRun.minX = -1.0;
    hRun.maxX =  6.0;
    hRun.minY = -1.0;
    hRun.maxY =  4.0;

    hRun.dt = 4.0e-4 / 4; //1.0e-3;
    hRun.tsn = 10000 * 4; //1000;
    hRun.ssi = 200 * 4;

    hGrid.oX = hRun.minX;
    hGrid.oY = hRun.minY;
    hGrid.size = 2.0 * hSmooth;
    hGrid.nX = (int) ((hRun.maxX - hRun.minX) / hGrid.size) +1;
    hGrid.nY = (int) ((hRun.maxY - hRun.minY) / hGrid.size) +1;


    printf("Dam break in a box \n");
    printf("Particles: %i \n", hPN);
}


void initFree() {

    int i, j, m, pi;
    double rho, c0, pmin;
    double dr;

    m = 1;
    rho = 1000.;
    c0 = 50.;
    pmin = -1.e12;

    hMatType[m] = 4;
    hMatProp[m][0] = rho;
    hMatProp[m][1] = c0;
    hMatProp[m][2] = pmin;

    dr = 0.01; // x4
    pi = 0;

    for (j = 0; j < 100; j++) {
        for (i = 0; i < 100; i++) {
            hPosX[pi] = i * dr + 0.0 * dr;
            hPosY[pi] = j * dr + 0.0 * dr;

            hVelX[pi] = 0.0;
            hVelY[pi] = 0.0;
            hMaterial[pi] = m;
            hDensity[pi] = rho; //+ (9.81 * rho / c0 / c0 * (50 - j) * dr);
            hEnergy[pi] = 0.0;
            hPressure[pi] = 1.0;
            pi++;
        }
    }
    
    hPN = pi;
    hSmooth = 1.2 * dr;
    hMass = rho * dr * dr;
    hSound = c0;

    hRun.minX = -0.5;
    hRun.maxX =  1.5;
    hRun.minY = -0.5;
    hRun.maxY =  1.5;

    hRun.dt = 0.5e-2; //1.0e-3;
    hRun.tsn = 3; //1000;
    hRun.ssi = 1;

    hGrid.oX = hRun.minX;
    hGrid.oY = hRun.minY;
    hGrid.size = 2.0 * hSmooth;
    hGrid.nX = (int) ((hRun.maxX - hRun.minX) / hGrid.size) +1;
    hGrid.nY = (int) ((hRun.maxY - hRun.minY) / hGrid.size) +1;


    printf("Freefall\n");
    printf("Particles: %i \n", hPN);
}



int iSort(int *array, int *perm, int n) {
    int i;
    static int* dummy = NULL;

    if (!dummy) dummy = (int *) malloc(MAXP * sizeof(int));

    for (i = 0; i < n; i++) dummy[i] = array[i];
    for (i = 0; i < n; i++) array[i] = dummy[perm[i]];

    return 0;
}

int fSort(float *array, int *perm, int n) {
    int i;
    static float* dummy = NULL;

    if (!dummy) dummy = (float *) malloc(MAXP * sizeof(float));

    for (i = 0; i < n; i++) dummy[i] = array[i];
    for (i = 0; i < n; i++) array[i] = dummy[perm[i]];

    return 0;
}


int sortArraysDevice(void) {
	int blocks, threads;
	
	threads = THREADS;
	blocks = (hPN + threads - 1) / threads;

    // Particles are re ordered
    
	kerSortInt <<< blocks, threads >>>
	(dIntDummy, dMaterial, dIndex);
    cutilSafeCall( hipMemcpy(dMaterial, dIntDummy, 
		(MAXP * sizeof(int)), hipMemcpyDeviceToDevice) );
	
	kerSortFloat <<< blocks, threads >>>
	(dFloatDummy, dPosX, dIndex);
    cutilSafeCall( hipMemcpy(dPosX, dFloatDummy, 
		(MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
	
	kerSortFloat <<< blocks, threads >>>
	(dFloatDummy, dPosY, dIndex);
    cutilSafeCall( hipMemcpy(dPosY, dFloatDummy, 
		(MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
	
	kerSortFloat <<< blocks, threads >>>
	(dFloatDummy, dVelX, dIndex);
    cutilSafeCall( hipMemcpy(dVelX, dFloatDummy, 
		(MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
	
	kerSortFloat <<< blocks, threads >>>
	(dFloatDummy, dVelY, dIndex);
    cutilSafeCall( hipMemcpy(dVelY, dFloatDummy, 
		(MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
	
	kerSortFloat <<< blocks, threads >>>
	(dFloatDummy, dDensity, dIndex);
    cutilSafeCall( hipMemcpy(dDensity, dFloatDummy, 
		(MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
	
	kerSortFloat <<< blocks, threads >>>
	(dFloatDummy, dEnergy, dIndex);
    cutilSafeCall( hipMemcpy(dEnergy, dFloatDummy, 
		(MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
	
	kerSortFloat <<< blocks, threads >>>
	(dFloatDummy, dPressure, dIndex);
    cutilSafeCall( hipMemcpy(dPressure, dFloatDummy, 
		(MAXP * sizeof(float)), hipMemcpyDeviceToDevice) );
	
    return 0;
}


int sortArraysHost(void) {

    // Particles are re ordered
    
    iSort(hHash, hIndex, hPN);
    iSort(hMaterial, hIndex, hPN);
    fSort(hPosX, hIndex, hPN);
    fSort(hPosY, hIndex, hPN);
    fSort(hVelX, hIndex, hPN);
    fSort(hVelY, hIndex, hPN);
    fSort(hDensity, hIndex, hPN);
    fSort(hEnergy, hIndex, hPN);
    fSort(hPressure, hIndex, hPN);
    
    return 0;
}


int indexCompare(const void *a, const void *b)
{
	int c, i1, i2;
	c = 0;
	i1 = *(int*)a;
	i2 = *(int*)b;
	if (hHash[i1] < hHash[i2]) c = -1;
	if (hHash[i1] > hHash[i2]) c = 1;
  return c;
}



int updateHashHost() {
	
    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */

	int ip, ix, iy, ic;
	
    for (ip = 0; ip < hPN; ip++) {
        ix = (int) ((hPosX[ip] - hGrid.oX) / hGrid.size);
        iy = (int) ((hPosY[ip] - hGrid.oY) / hGrid.size);
		ic = ix + iy * hGrid.nX;
		
		hHash[ip] = ic;
		hIndex[ip] = ip;
	}
	
	return 0;
}

int updateSetsHost() {
	
    /**
     * \brief Update particles
     *
     * \date Jan 6, 2010
     * \author Luca Massidda
     */
	
	int ip;
	
	hSetStart[hHash[0]] = 0;
	if (hHash[0] != hHash[1]) hSetStop[hHash[0]] = 1;

    for (ip = 1; ip < hPN -1; ip++) {
		if (hHash[ip] != hHash[ip -1]) hSetStart[hHash[ip]] = ip;
		if (hHash[ip] != hHash[ip +1]) hSetStop[hHash[ip]] = ip +1;
	}
	
	if (hHash[hPN -1] != hHash[hPN -2]) hSetStart[hHash[hPN -1]] = hPN -1;
	hSetStop[hHash[hPN -1]] = hPN;
	
	return 0;
}


int updateListHost(void) {
	int ip, ic, ix, iy, il, i, j, jp, jc, np;
	float dx, dy, dr;
	
    // Particles list is filled
    for (ip = 0; ip < hPN; ip++) {
		for (il = 0; il < MAXN; il++) {
			hList[ip * MAXN + il] = ip;
		}
		
        ix = (int) ((hPosX[ip] - hGrid.oX) / hGrid.size);
        iy = (int) ((hPosY[ip] - hGrid.oY) / hGrid.size);
		ic = ix + iy * hGrid.nX;
		
		np = 0;
        for (j = -1; j <= 1; j++) {
            for (i = -1; i <= 1; i++) {
				jc = ic + i + j * hGrid.nX;
				
				for (jp = hSetStart[jc]; jp < hSetStop[jc]; jp++) {
					dx = hPosX[ip] - hPosX[jp];
                    dy = hPosY[ip] - hPosY[jp];
                    dr = sqrtf(dx * dx + dy * dy);
					
					if ((dr < 2.0 * hSmooth) && (np < MAXN)) {
						hList[ip * MAXN + np] = jp;
						np++;
					}
				}
			}
		}
		
		
	}
	
	return 0;
}


int neighbourListDevice() {
	int blocks, threads;
	
	blocks = (hPN + THREADS - 1) / THREADS;
	threads = THREADS;
	
	updateHashDevice <<< blocks, threads >>>
	(dGrid, dPosX, dPosY, dHash, dIndex);
	
	cudppSort(hSortHandle, dHash, dIndex, 24, hPN);
	
	sortArraysDevice();
	
	cutilSafeCall( hipMemset(dSetStart, 0, dGrid.nX*dGrid.nY * sizeof(int)));
	cutilSafeCall( hipMemset(dSetStop, 0, dGrid.nX*dGrid.nY * sizeof(int)));
	
	updateSetsDevice <<< blocks, threads >>>
	(dSetStart, dSetStop, dHash);
	
	updateListDevice <<< blocks, threads >>>
	(dList, dSetStart, dSetStop, dGrid, dPosX, dPosY);
	
	return 0;
}

int neighbourListHost() {
	
	updateHashHost();
	
	qsort(hIndex, hPN, sizeof(int), indexCompare);
	
	sortArraysHost();
	
	updateSetsHost();
	
	updateListHost();
	
	return 0;
}

int RKstepDevice(float alpha) {
	int blocks, threads;
	
	blocks = (hPN + THREADS - 1) / THREADS;
	threads = THREADS;
	
	// External forces
	updateForcesDevice <<< blocks, threads >>>
	(dMaterial, dVelDotX, dVelDotY, dDensityDot, dEnergyDot);
	
	// Calculate particle interactions
	balanceMassMomentumDevice <<< blocks, threads >>>
	(dList, dPosX, dPosY, dVelX, dVelY, dDensity, dPressure, 
	dDensityDot, dVelDotX, dVelDotY);
	
	balanceEnergyDevice <<< blocks, threads >>>
	(dPressure, dDensity, dDensityDot, dEnergyDot);
	
	// Update particles
	updateParticlesDevice  <<< blocks, threads >>> 
	(dMaterial, dVelDotX, dVelDotY, dDensityDot, dEnergyDot, alpha,
	dPosX0, dPosY0, dVelX0, dVelY0, dDensity0, dEnergy0,
	dPosX, dPosY, dVelX, dVelY, dDensity, dEnergy, dPressure);
	
	return 0;
}

int RKstepHost(float alpha) {
	
	// External forces
	updateForcesHost();
		
	// Calculate particle interactions
	balanceMassMomentumHost();
	
	balanceEnergyHost();
	
	// Update particles
	updateParticlesHost(alpha);
	
	return 0;
}

int RKintegrateDevice(void) {

    /**
     * \brief Runge Kutta 3rd order time integration
     *
     * Integrate the Navier Stokes equations in time with the
     * Total Variation Diminishing Runge-Kutta algorithm of the 3rd order
     *
     * \date Dec 20, 2010
     * \author Luca Massidda
     */

    int ts;

    // Output
    //outputCase();
	
	copyHostToDevice();

    // TIME CYCLE
    for (ts = 0; ts <= hRun.tsn; ts++) {
		
        // Output data
        if ((ts % hRun.ssi) == 0) {
            printf("Saving time: %g \n", ts * hRun.dt);
			copyDeviceToHost();
            printData();
            //outputData(ts / hRun.ssi);
            outputVTK(ts / hRun.ssi);
        }
		
		// Calculate neighbour list
		neighbourListDevice();
		
		/*
		copyDeviceToHost();
		updateHash();
		sortArraysHost();
		for (int i = 0; i < hGrid.nX*hGrid.nY; i++) {
			hSetStart[i] = 0;
			hSetStop[i] = 0;
		}
		updateGrid();
		updateList();
		copyHostToDevice();
		*/
		
		// Save initial condition
		backupData();
		
        // Step 1
		RKstepDevice(1.0);
		
        // Step 2
		RKstepDevice(1.0 / 4.0);
		
        // Step 3
		RKstepDevice(2.0 / 3.0);
	}
	
	cutilSafeCall( hipDeviceReset() );
	
	return 0;
}

int RKintegrateHost(void) {

    /**
     * \brief Runge Kutta 3rd order time integration
     *
     * Integrate the Navier Stokes equations in time with the
     * Total Variation Diminishing Runge-Kutta algorithm of the 3rd order
     *
     * \date Dec 20, 2010
     * \author Luca Massidda
     */

    int ts;
	
    // Output
    //outputCase();
	
    // TIME CYCLE
    for (ts = 0; ts <= hRun.tsn; ts++) {
		
        // Output data
        if ((ts % hRun.ssi) == 0) {
            printf("Saving time: %g \n", ts * hRun.dt);
            printData();
            //outputData(ts / hRun.ssi);
            outputVTK(ts / hRun.ssi);
        }
        
		// Calculate neighbour list
		neighbourListHost();
		
		// Save initial condition
		backupDataHost();
		
        // Step 1
		RKstepHost(1.0);
		
        // Step 2
		RKstepHost(1.0 / 4.0);
		
        // Step 3
		RKstepHost(2.0 / 3.0);
	}
	
	return 0;
}

int check() {
    float *rVelDotX, *rVelDotY, *rDensityDot;
    float sumVelDotX, maxVelDotX;
    float sumVelDotY, maxVelDotY;
    float sumDensityDot, maxDensityDot;
    FILE *stream;
    
    rVelDotX = (float *) malloc(MAXP * sizeof(float));
    rVelDotY = (float *) malloc(MAXP * sizeof(float));
    rDensityDot = (float *) malloc(MAXP * sizeof(float));
	
	for (int i = 0; i < hPN; i++) {
		hPressure[i] = 1000.0 * (sin(10*M_PI * hPosX[i]) + hPosY[i]*hPosY[i]);
		hVelX[i] = sin(10*M_PI * hPosX[i]);
		hVelY[i] = hPosY[i]*hPosY[i];
	}
	
	int blocks = (hPN + THREADS - 1) / THREADS;
	copyHostToDevice();
	
	neighbourListDevice();
	
	// Calculate particle interactions
	balanceMassMomentumDevice <<< blocks, THREADS >>>
	(dList, dPosX, dPosY, dVelX, dVelY, dDensity, dPressure, 
	dDensityDot, dVelDotX, dVelDotY);
	
	copyDeviceToHost();
	
	for (int i = 0; i < hPN; i++) {
		rVelDotX[i] = -1 * 10*M_PI * cos(10*M_PI * hPosX[i]);
		rVelDotY[i] = -1 * 2.0 *hPosY[i];
		rDensityDot[i] = -1000.0 * (10*M_PI * cos(10*M_PI * hPosX[i]) + 2.0 *hPosY[i]);
	}
	
	for (int i = 0; i < hPN; i++) {
		if ((hPosX[i] < 1*hGrid.size) ||
			(hPosY[i] < 1*hGrid.size) ||
			(hPosX[i] > 1.0 - 1*hGrid.size) ||
			(hPosY[i] > 1.0 - 1*hGrid.size)) {
				hVelDotX[i] = rVelDotX[i];
				hVelDotY[i] = rVelDotY[i];
				hDensityDot[i] = rDensityDot[i];
		}
	}
	
	sumVelDotX = 0.0;
	maxVelDotX = 0.0;
	sumVelDotY = 0.0;
	maxVelDotY = 0.0;
	sumDensityDot = 0.0;
	maxDensityDot = 0.0;
	
	for (int i = 0; i < hPN; i++) {
		sumVelDotX += abs(rVelDotX[i] - hVelDotX[i]);
		sumVelDotY += abs(rVelDotY[i] - hVelDotY[i]);
		sumDensityDot += abs(rDensityDot[i] - hDensityDot[i]);
		
		if (abs(rVelDotX[i]) > maxVelDotX) 
			maxVelDotX = abs(rVelDotX[i]);
		if (abs(rVelDotY[i]) > maxVelDotY) 
			maxVelDotY = abs(rVelDotY[i]);
		if (abs(rDensityDot[i]) > maxDensityDot) 
			maxDensityDot = abs(rDensityDot[i]);		
	}
	
	printf("VelDotX errors %f%% \n", 100 * sumVelDotX/hPN/maxVelDotX);
	printf("VelDotY errors %f%% \n", 100 * sumVelDotY/hPN/maxVelDotY);
	printf("DensityDot errors %f%% \n", 100 * sumDensityDot/hPN/maxDensityDot);
	
    stream = fopen("debug.txt", "w");
    for (int i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e\n", hDensityDot[i],
                hVelDotX[i], hVelDotY[i]);
    fclose(stream);

    stream = fopen("reference.txt", "w");
    for (int i = 0; i < hPN; i++)
        fprintf(stream, "%+14.8e %+14.8e %+14.8e\n", rDensityDot[i],
                rVelDotX[i], rVelDotY[i]);
    fclose(stream);
    
	for (int i = 0; i < hPN; i++) {
		hVelX[i] = hVelDotX[i];
		hVelY[i] = hVelDotY[i];
		hDensity[i] = hDensityDot[i];
	}
	
	hRun.tsn = 1;
	outputCase();
	outputData(0);
	
	return 0;
}


int main() {
    /**
     * \brief armando2D v2.0
     *
     * An SPH code for non stationary fluid dynamics.
     * This is the reviewed and improved C version of Armando v1.0
     * developed at CERN in 2008
     *
     * \date Oct 20, 2010
     * \author Luca Massidda
     */

    initHost();

    initDamBreak();
    //initFree();

    //initDevice();
    //initCUDPP();
    
	//RKintegrateDevice();
	RKintegrateHost();
	//check();
	
    return 0;
}
